#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////////////////
//  PVR Texture Compressor														//
//  Author: Chirantan Ekbote													//
//	Mailto: ekbote.1@osu.edu													//
//																				//
//  Harvard School of Engineering and Applied Sciences							//
//  The Ohio State University													//
//																				//
//	PVRTexCUDA.cu - Contains CUDA compression code								//
//																				//
//  This is a GPU implementation of a compressor for the PVR format as			//
//  described by Simon Fenney in his paper Texture Compression using			//
//  Low-Frequency Signal Modulation.											//
//																				//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//									File Includes	 							//
//////////////////////////////////////////////////////////////////////////////////
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cutil_inline.h>
#include <stdio.h>
#include <limits.h>
//////////////////////////////////////////////////////////////////////////////////
//									File Includes	 							//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//							   Preprocessor Definitions	 						//
//////////////////////////////////////////////////////////////////////////////////
#define TWO_BPP
#define USE_PIXEL_UPDATE
#define USE_SVD
//#define USE_CHOLESKY
//#define USE_JAMA_SVD
//#define GET_RMS_ERROR
//#define GET_SNR
//#define DECOMPRESS_PVR
#define TWO_BY_TWO
#define BLOCK_WIDTH				16
#define BLOCK_HEIGHT			16
#define FILTER_LENGTH			2
#define HALF_FILTER_LENGTH		FILTER_LENGTH * 0.5f
#define RED_SHIFT				16
#define GREEN_SHIFT				8
#define	BLUE_SHIFT				0
#define RSHIFT_16BPP			3
#define LSHIFT_16BPP			5
#define ALPHA_MASK				0xFF000000
#define RED_MASK				0x00FF0000
#define GREEN_MASK				0x0000FF00
#define BLUE_MASK				0x000000FF
#define ONE_SIXTEENTH           0.0625f
#define ONE_EIGHTH				0.125f
#define ONE_FOURTH				0.25f
#define THREE_EIGHTHS			0.375f
#define FIVE_EIGHTHS			0.625f
#define NUM_OPTIMIZATION_PASSES	20
#define EPSILON					10e-10
//////////////////////////////////////////////////////////////////////////////////
//							   Preprocessor Definitions	 						//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//								  Macro Definitions		 						//
//////////////////////////////////////////////////////////////////////////////////
#define SQR(x)					((x) * (x))
#define CLAMP(x, a, b)			(min((max((x), (a))), (b)))
#define MAKE_RED_PIXEL(x)		(((x) & RED_MASK)>>RED_SHIFT)
#define MAKE_GREEN_PIXEL(x)		(((x) & GREEN_MASK)>>GREEN_SHIFT)
#define MAKE_BLUE_PIXEL(x)		(((x) & BLUE_MASK)>>BLUE_SHIFT)
#define MAKE_ARGB(r, g, b)		(ALPHA_MASK | ((r)<<RED_SHIFT) | \
                                ((g)<<GREEN_SHIFT) | ((b)<<BLUE_SHIFT))
//////////////////////////////////////////////////////////////////////////////////
//								  Macro Definitions		 						//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//									   Misc.									//
//////////////////////////////////////////////////////////////////////////////////
// CUDA texture declarations
texture<unsigned int, hipTextureType2D, hipReadModeElementType> texRef;
texture<unsigned int, hipTextureType2D, hipReadModeElementType> origRef;
texture<int, hipTextureType2D, hipReadModeElementType> redCurrentRef;
texture<int, hipTextureType2D, hipReadModeElementType> greenCurrentRef;
texture<int, hipTextureType2D, hipReadModeElementType> blueCurrentRef;
texture<float, hipTextureType2D, hipReadModeElementType> modRef;
texture<float, hipTextureType1D, hipReadModeElementType> filterRef;
#ifdef TWO_BY_TWO
texture<float, hipTextureType1D, hipReadModeElementType> svdMatRefTL;
texture<float, hipTextureType1D, hipReadModeElementType> svdMatRefTR;
texture<float, hipTextureType1D, hipReadModeElementType> svdMatRefBL;
texture<float, hipTextureType1D, hipReadModeElementType> svdMatRefBR;
#else
texture<float, hipTextureType1D, hipReadModeElementType> svdMatRef;
#endif // TWO_BY_TWO

// various wavelet filters. remember to update the FILTER_LENGTH macro if you
// choose to use a different one

// bior wavelet filter
//static const float wavelet_filter[] = {
//	-0.00488281250000000000,
//	0.00976562500000000000,
//	0.03320312500000000000,
//	-0.07617187500000000000,
//	-0.12011718750000000000,
//	0.31640625000000000000,
//	0.68359375000000000000,
//	0.31640625000000000000,
//	-0.12011718750000000000,
//	-0.07617187500000000000,
//	0.03320312500000000000,
//	0.00976562500000000000,
//	-0.00488281250000000000,
//};

// 2nd bior wavelet filter
//static const float wavelet_filter[] = {
//-0.0625,
//0.06250,
//0.50000,
//0.50000,
//0.06250,
//-0.0625
//};

// my wavelet filter
//static const float wavelet_filter[] = {
//-0.2500,
//0.50000,
//0.50000,
//0.50000,
//-0.2500
//};

// daubechies wavelet filter
//static const float wavelet_filter[] = {
//	0.02692517479416041400,
//	0.17241715192471294000,
//	0.42767453217028290000,
//	0.46477285717277800000,
//	0.09418477475112015100,
//	-0.20737588089628295000,
//	-0.06847677451090331000,
//	0.10503417113713563000,
//	0.02172633772990401800,
//	-0.04782363205881859400,
//	0.00017744640673182261,
//	0.01581208292613723100,
//	-0.00333981011324138060,
//	-0.00302748028715121120,
//	0.00130648364017893680,
//	0.00016290733600968354,
//	-0.00017816487954739422,
//	0.00002782275679290904
//};

// simple wavelet filter
static float wavelet_filter[] = {0.5f, 0.5f};

// weight matrix to be combined with modulation data in the optimization step
#ifdef TWO_BPP
#define ROW(n)	1*n, 2*n, 3*n, 4*n, 5*n, 6*n, 7*n, 8*n, 7*n, 6*n, 5*n, 4*n, 3*n, 2*n, 1*n
#else
#define ROW(n)	1*n,	2*n,	3*n,	4*n,	3*n,	2*n,	1*n
#endif // TWO_BPP

#ifdef TWO_BY_TWO

#ifdef TWO_BPP
#define SVD_FACTOR_X            ONE_SIXTEENTH
#define SVD_FACTOR_Y			ONE_EIGHTH
#define SVD_OFFSET_X            8
#define SVD_OFFSET_Y			4
#define SVD_MAT_WIDTH			8
#define SVD_MAT_HEIGHT			253
#define SVD_DIM_X               23
#define SVD_DIM_Y				11
static const float MwTL[] = 
{
	ROW(0.03125f), 0, 0, 0, 0, 0, 0, 0, 0,	// 1/32
	ROW(0.0625f), 0, 0, 0, 0, 0, 0, 0, 0,	// 2/32
	ROW(0.09375f), 0, 0, 0, 0, 0, 0, 0, 0,	// 3/32
	ROW(0.125f), 0, 0, 0, 0, 0, 0, 0, 0,    // 4/32
	ROW(0.09375f), 0, 0, 0, 0, 0, 0, 0, 0,	// 3/32
	ROW(0.0625f), 0, 0, 0, 0, 0, 0, 0, 0,	// 2/32
	ROW(0.03125f), 0, 0, 0, 0, 0, 0, 0, 0,	// 1/32
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0
};
static const float MwTR[] = 
{
    0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.03125f),  // 1/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.0625f),   // 2/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.09375f),  // 3/32
	0, 0, 0, 0, 0, 0, 0, 0, ROW(0.125f),    // 4/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.09375f),  // 3/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.0625f),   // 2/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.03125f),  // 1/32
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0
};
static const float MwBL[] = 
{
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0.03125f), 0, 0, 0, 0, 0, 0, 0, 0,	// 1/32
	ROW(0.0625f), 0, 0, 0, 0, 0, 0, 0, 0,	// 2/32
	ROW(0.09375f), 0, 0, 0, 0, 0, 0, 0, 0,	// 3/32
	ROW(0.125f), 0, 0, 0, 0, 0, 0, 0, 0,    // 4/32
	ROW(0.09375f), 0, 0, 0, 0, 0, 0, 0, 0,	// 3/32
	ROW(0.0625f), 0, 0, 0, 0, 0, 0, 0, 0,	// 2/32
	ROW(0.03125f), 0, 0, 0, 0, 0, 0, 0, 0	// 1/32
};
static const float MwBR[] = 
{
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0, 0, 0, 0, 0,
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.03125f),  // 1/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.0625f),   // 2/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.09375f),  // 3/32
	0, 0, 0, 0, 0, 0, 0, 0, ROW(0.125f),    // 4/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.09375f),  // 3/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.0625f),   // 2/32
	0, 0, 0, 0, 0, 0, 0, 0,	ROW(0.03125f)   // 1/32
};

#else

#define SVD_FACTOR_X            ONE_EIGHTH
#define SVD_FACTOR_Y			ONE_EIGHTH
#define SVD_OFFSET_X            4
#define SVD_OFFSET_Y			4
#define SVD_MAT_WIDTH			8
#define SVD_MAT_HEIGHT			121
#define SVD_DIM_X               11
#define SVD_DIM_Y				11
static const float MwTL[] = 
{
	ROW(0.0625f), 0, 0, 0, 0,	// 1/16
	ROW(0.125f), 0, 0, 0, 0,	// 2/16
	ROW(0.1875f), 0, 0, 0, 0,	// 3/16
	ROW(0.25f), 0, 0, 0, 0,		// 4/16
	ROW(0.1875f), 0, 0, 0, 0,	// 3/16
	ROW(0.125f), 0, 0, 0, 0,	// 2/16
	ROW(0.0625f), 0, 0, 0, 0,	// 1/16
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0
};
static const float MwTR[] = 
{
	0, 0, 0, 0, ROW(0.0625f),	// 1/16
	0, 0, 0, 0, ROW(0.125f),	// 2/16
	0, 0, 0, 0, ROW(0.1875f),	// 3/16
	0, 0, 0, 0, ROW(0.25f),		// 4/16
	0, 0, 0, 0, ROW(0.1875f),	// 3/16
	0, 0, 0, 0, ROW(0.125f),	// 2/16
	0, 0, 0, 0, ROW(0.0625f),	// 1/16
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0
};
static const float MwBL[] = 
{
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0.0625f), 0, 0, 0, 0,	// 1/16
	ROW(0.125f), 0, 0, 0, 0,	// 2/16
	ROW(0.1875f), 0, 0, 0, 0,	// 3/16
	ROW(0.25f), 0, 0, 0, 0,		// 4/16
	ROW(0.1875f), 0, 0, 0, 0,	// 3/16
	ROW(0.125f), 0, 0, 0, 0,	// 2/16
	ROW(0.0625f), 0, 0, 0, 0,	// 1/16
};
static const float MwBR[] = 
{
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	ROW(0), 0, 0, 0, 0,
	0, 0, 0, 0, ROW(0.0625f),	// 1/16
	0, 0, 0, 0, ROW(0.125f),	// 2/16
	0, 0, 0, 0, ROW(0.1875f),	// 3/16
	0, 0, 0, 0, ROW(0.25f),		// 4/16
	0, 0, 0, 0, ROW(0.1875f),	// 3/16
	0, 0, 0, 0, ROW(0.125f),	// 2/16
	0, 0, 0, 0, ROW(0.0625f)	// 1/16
};
#endif  // TWO_BPP

#else

#ifdef TWO_BPP
#define SVD_FACTOR_X            ONE_EIGHTH
#define SVD_FACTOR_Y			ONE_FOURTH
#define SVD_OFFSET_X            8
#define SVD_OFFSET_Y			4
#define SVD_MAT_WIDTH			2
#define SVD_MAT_HEIGHT			105
#define SVD_DIM_X               15
#define SVD_DIM_Y				7
static const float Mw[] = 
{
	ROW(0.03125f),	// 1/32
	ROW(0.0625f),	// 2/32
	ROW(0.09375f),	// 3/32
	ROW(0.125f),    // 4/32
	ROW(0.09375f),	// 3/32
	ROW(0.0625f),	// 2/32
	ROW(0.03125f)	// 1/32
};
#else
#define SVD_FACTOR_X			ONE_FOURTH
#define SVD_FACTOR_Y			ONE_FOURTH
#define SVD_OFFSET_X			4
#define SVD_OFFSET_Y			4
#define SVD_MAT_WIDTH			2
#define SVD_MAT_HEIGHT			49
#define SVD_DIM_X				7
#define SVD_DIM_Y				7
static const float Mw[] = 
{
	ROW(0.0625f),	// 1/16
	ROW(0.125f),	// 2/16
	ROW(0.1875f),	// 3/16
	ROW(0.25f),		// 4/16
	ROW(0.1875f),	// 3/16
	ROW(0.125f),	// 2/16
	ROW(0.0625f)	// 1/16
};
#endif // TWO_BPP
#endif // TWO_BY_TWO
// pointer swap method
static inline void swap(void **x, void **y) {
	void *t = *x;
	*x = *y;
	*y = t;
}

//////////////////////////////////////////////////////////////////////////////////
//	Calculate the root mean squared error of the image that is decoded from the	//
//	compressed data. This kernel calculates the total error for the red, green,	//
//	and blue channels of a single pixel and stores it in an array.				//
//																				//
//	orig		Pointer to the pixel data for the original image				//
//	a			Pointer to the upscaled pixel data for the final A image		//
//	b			Pointer to the upscaled pixel data for the final B image		//
//	out			Pointer to the array where the total error for the current 		//
//				pixel is stored													//
//	width		The width, in pixels, of the original image						//
//																				//
//////////////////////////////////////////////////////////////////////////////////
#ifdef GET_RMS_ERROR
__global__ void rms_error(unsigned int *orig, unsigned int *a, unsigned int *b,
						  unsigned int *out,  int width) {
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = (width * y) + x;

	unsigned int opixel = orig[idx];
	unsigned int apixel = a[idx];
	unsigned int bpixel = b[idx];
	float modbit = tex2D(modRef, x, y);
	float r_modbit = 1.0f - modbit;
	int c_red = ((float)MAKE_RED_PIXEL(bpixel)*modbit + 
		(float)MAKE_RED_PIXEL(apixel)*r_modbit);
	int c_green = ((float)MAKE_GREEN_PIXEL(bpixel)*modbit + 
		(float)MAKE_GREEN_PIXEL(apixel)*r_modbit);
	int c_blue = ((float)MAKE_BLUE_PIXEL(bpixel)*modbit + 
		(float)MAKE_BLUE_PIXEL(apixel)*r_modbit);

	out[idx] = SQR(((float)MAKE_RED_PIXEL(opixel) - c_red)) + 
			   SQR(((float)MAKE_GREEN_PIXEL(opixel) - c_green)) + 
			   SQR(((float)MAKE_BLUE_PIXEL(opixel) - c_blue));
}
#endif // GET_RMS_ERROR

#ifdef DECOMPRESS_PVR
__global__ void decompress(unsigned int *a, unsigned int *b, 
						   unsigned int *out, float *mod, int width) {
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = (width * y) + x;

	unsigned int apixel = a[idx];
	int a_red = MAKE_RED_PIXEL(apixel);
	int a_green = MAKE_GREEN_PIXEL(apixel);
	int a_blue = MAKE_BLUE_PIXEL(apixel);

	unsigned int bpixel = b[idx];
	int b_red = MAKE_RED_PIXEL(bpixel);
	int b_green = MAKE_GREEN_PIXEL(bpixel);
	int b_blue = MAKE_BLUE_PIXEL(bpixel);

	float modbit = mod[idx];
	float r_modbit = 1.0f - modbit;

	int c_red = ((float)b_red*modbit + (float)a_red*r_modbit);
	int c_green = ((float)b_green*modbit + (float)a_green*r_modbit);
	int c_blue = ((float)b_blue*modbit + (float)a_blue*r_modbit);

	out[idx] = MAKE_ARGB(c_red, c_green, c_blue);

}
#endif // DECOMPRESS_PVR

__global__ void encode_texture(const unsigned int* A, unsigned int *B, float *mod, 
							   int width, bool hasAlpha, unsigned int *out, 
							   unsigned int *mode, int outWidth, int outHeight) {
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned outy = y, outx = x;
	//int idx = (width * CLAMP(y, 15, outHeight-15)) + (CLAMP(x, 15, outWidth-15));
	/*if (x < 10) {
		outx = 15;
		outy = 15;
	} else if (x > outWidth-10) {
		outx = 80;
		outy = 45;
	} 
	if (y < 10) {
		outy = 15;
		outx = 15;
	} else if (y > outHeight-10) {
		outy = 45;
		outx = 80;
	}*/ 
	int idx = (width*outy) + outx;

	// Interleave lower 16 bits of x and y, so the bits of y
	// are in the even positions and bits from x in the odd;
	// outIdx is the resulting 32-bit Morton Number. 
	//unsigned int a = (2*x >= outWidth) ? 2*x - outWidth : 2*x;
	unsigned int a = 2*x;
	unsigned int b = y;

	a = (a | (a << 8)) & 0x00FF00FF;
	a = (a | (a << 4)) & 0x0F0F0F0F;
	a = (a | (a << 2)) & 0x33333333;
	a = (a | (a << 1)) & 0x55555555;

	b = (b | (b << 8)) & 0x00FF00FF;
	b = (b | (b << 4)) & 0x0F0F0F0F;
	b = (b | (b << 2)) & 0x33333333;
	b = (b | (b << 1)) & 0x55555555;

	// multiply by two because we encode 64-bit blocks not 32-bit blocks
	//int outIdx = 2 * CLAMP((a | (b << 1)), 0, outWidth*outHeight);
	// TODO: Fix this, currently only works if dimnesion is 2x as big
	int outIdx = (a | (b << 1));
	//outIdx += (2*x >= outWidth) ? (outWidth*outHeight) : 0;

	unsigned int apixel = A[idx];
	unsigned int bpixel = B[idx];
	unsigned int color = 0, modbits = 0;

	if (hasAlpha == false) {
		// encode color B
		color = 1;
		color = ((color<<LSHIFT_16BPP) | (MAKE_RED_PIXEL(bpixel)>>RSHIFT_16BPP));
		color = ((color<<LSHIFT_16BPP) | (MAKE_GREEN_PIXEL(bpixel)>>RSHIFT_16BPP));
		color = ((color<<LSHIFT_16BPP) | (MAKE_BLUE_PIXEL(bpixel)>>RSHIFT_16BPP));
		// encode color A
		color = ((color<<1) | 1);
		color = ((color<<LSHIFT_16BPP) | (MAKE_RED_PIXEL(apixel)>>RSHIFT_16BPP));
		color = ((color<<LSHIFT_16BPP) | (MAKE_GREEN_PIXEL(apixel)>>RSHIFT_16BPP));
		color = ((color<<(LSHIFT_16BPP-1)) | (MAKE_BLUE_PIXEL(apixel)>>(RSHIFT_16BPP+1)));

		// modulation mode 0
#ifdef TWO_BPP
        int modulation_mode = 1;//mode[idx];
        color = ((color<<1) | modulation_mode);
#else
		color = ((color<<1) | 0);
#endif  // TWO_BPP

		// encode modulation bits
#ifdef TWO_BPP
        int i, j, checker;
		float mbit;
		for (i = 3; i >= 0; i--) {
            if (modulation_mode == 1) {
                checker = !(i & 1);
                for (j = 3; j >= 0; j--) {
                    mbit = tex2D(modRef, (8*x) + (2 * j) + checker, i + (4*y));
                    if (mbit < 0.05f) {
                        modbits = (modbits<<2);
                    } else if (mbit < 0.5f) {
                        modbits = ((modbits<<2) | (0x1));
                    } else if (mbit < 0.95f) {
                        modbits = ((modbits<<2) | (0x2));
                    } else {
                        modbits = ((modbits<<2) | (0x3));
                    }
                }
            } else {
                for (j = 7; j >= 0; j--) {
                    mbit = tex2D(modRef, (8*x) + (2 * j) + checker, i + (4*y));
                    if (mbit < 0.5f) {
                        modbits = (modbits<<1);
                    } else {
                        modbits = ((modbits<<1) | (0x1));
                    }
                }
            }
		}
#else
		int i, j;
		float mbit;
		for (i = 3; i >= 0; i--) {
			for (j = 3; j >= 0; j--) {
				mbit = tex2D(modRef, j + (4*x), i + (4*y));
				if (mbit < 0.05f) {
					modbits = (modbits<<2);
				} else if (mbit < 0.5f) {
					modbits = ((modbits<<2) | (0x1));
				} else if (mbit < 0.95f) {
					modbits = ((modbits<<2) | (0x2));
				} else {
					modbits = ((modbits<<2) | (0x3));
				}
			}
		}
#endif

		out[outIdx] = modbits;
		out[outIdx+1] = color;
	} else {
		// TODO: implement this
	}
}
//////////////////////////////////////////////////////////////////////////////////
//									   Misc.									//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//								  Optimization									//
//////////////////////////////////////////////////////////////////////////////////
__device__ float pythag(float a, float b) {
	float absa = fabsf(a);
	float absb = fabsf(b);
	if (absa > absb) return absa*sqrtf(1.0f+SQR((absb/absa)));
	else return (absb < EPSILON ? 0.0f : absb*sqrtf(1.0f+SQR(absa/absb)));
}

__device__ float sign(float a, float b) {
	return b >= 0.0f ? ( a >= 0.0f ? a : -a) : ( a >= 0.0f ? -a : a);
}

#ifndef TWO_BY_TWO
//////////////////////////////////////////////////////////////////////////////////
//	Calculate the Moore-Penrose pseudo-inverse of the weight matrix and use		//
//	it to compute the updated pair of A and B representative pixels.			//
//																				//
//	candidateA	Pointer to the output location of the A image					//
//	candidateB	Pointer to the output location of the B image					//
//	oldA		Pointer to the A image computed in the previous iteration		//
//	oldB		Pointer to the B image computed in the previous iteration		//
//	width		The width, in pixels, of the image to be compressed				//
//	err			Pointer to memory where errors are reported						//
//																				//
//////////////////////////////////////////////////////////////////////////////////
__global__ void moore_penrose_optimize(unsigned int *candidateA, unsigned int *candidateB,
									   unsigned int *oldA, unsigned int *oldB, 
									   int width, int height, int *err) {
	int thready = blockDim.y * blockIdx.y + threadIdx.y;
	int threadx = blockDim.x * blockIdx.x + threadIdx.x;
	const int  tIdx = (width*thready) + threadx;
	int red,red0=0,red1=0,green,green0=0,green1=0,blue,blue0=0,blue1=0;
	int x_offset = SVD_OFFSET_X*threadx-1, y_offset = SVD_OFFSET_Y*thready-1;
	int x, y, pixelx, pixely, index;
	float A00=0,A01=0,A10=0,A11=0,InverseA00,InverseA01,InverseA10,InverseA11;
	float a0, a1, modbit, dist, det;

	// Fetch the weight matrix for the 7x7 optimization window of the current
	// pair of representative values
	for (y = 0; y < SVD_DIM_Y; y++) {
		for (x = 0; x < SVD_DIM_X; x++) {
			index = y*SVD_DIM_X + x;
			pixelx = x_offset + x;
			pixely = y_offset + y;

			// fetch all the necessary values
			modbit = tex2D(modRef, CLAMP(pixelx, 0, width - 2), CLAMP(pixely, 0, height-2));
			red = tex2D(redCurrentRef, CLAMP(pixelx, 0, width-2), CLAMP(pixely, 0, height-2));
			green = tex2D(greenCurrentRef, CLAMP(pixelx, 0, width-2), CLAMP(pixely, 0, height-2));
			blue = tex2D(blueCurrentRef, CLAMP(pixelx, 0, width-2), CLAMP(pixely, 0, height-2));
			dist = tex1Dfetch(svdMatRef, index);
			a0 = dist*(1.0f - modbit);
			a1 = dist*modbit;	

			// A = TransposeA * A
			A00 += a0*a0;
			A01 += a0*a1;
			A10 += a1*a0;
			A11 += a1*a1;

			// colors = TransposeA * color
			red0 += a0*red;
			red1 += a1*red;

			green0 += a0*green;
			green1 += a1*green;

			blue0 += a0*blue;
			blue1 += a1*blue;
		}
	}

	// since we only have a 2x2 matrix, manually calculate inverse
	det = 1.0f / (A00*A11 - A01*A10);
	InverseA00 = det * A11;
	InverseA01 = -1 * det * A01;
	InverseA10 = -1 * det * A10;
	InverseA11 = det * A00;

	// Calculate new representative colors by using a "fix the error" approach.
	// We multiply the pseudo-inverse weight matrix by the difference between the
	// current and original images to get an "update" that is applied to each
	// representative value. We also limit how big the update can be to avoid
	// colors flying out of bounds.
	unsigned int oldColor;

	// A representative
	oldColor = oldA[tIdx];
	red = InverseA00*(float)red0 + InverseA01*(float)red1;
	green = InverseA00*(float)green0 + InverseA01*(float)green1;
	blue = InverseA00*(float)blue0 + InverseA01*(float)blue1;
#ifdef USE_PIXEL_UPDATE
	candidateA[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + CLAMP((int)red, -16, 16), 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + CLAMP((int)green, -16, 16), 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + CLAMP((int)blue, -16, 16), 0, 255));
#else
	candidateA[tIdx] = MAKE_ARGB(CLAMP(red, 0, 255), CLAMP(green, 0, 255), CLAMP(blue, 0, 255));
#endif // USE_PIXEL_UPDATE
	oldA[tIdx] = candidateA[tIdx];

	// B representative
	oldColor = oldB[tIdx];
	red = InverseA10*(float)red0 + InverseA11*(float)red1;
	green = InverseA10*(float)green0 + InverseA11*(float)green1;
	blue = InverseA10*(float)blue0 + InverseA11*(float)blue1;
#ifdef USE_PIXEL_UPDATE
	candidateB[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + CLAMP((int)red, -16, 16), 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + CLAMP((int)green, -16, 16), 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + CLAMP((int)blue, -16, 16), 0, 255));
#else
	candidateB[tIdx] = MAKE_ARGB(CLAMP(red, 0, 255), CLAMP(green, 0, 255), CLAMP(blue, 0, 255));
#endif // USE_PIXEL_UPDATE
	oldB[tIdx] = candidateB[tIdx];
}
#endif // TWO_BY_TWO
//////////////////////////////////////////////////////////////////////////////////
//	Calculates the singular value decomposition of the weight matrix and uses	//
//	it to compute teh pseudo-inverse. Then multiplies the pseudo-niverse by		//
//	the current color values to get the new values.								//
//	DOCUMENTATION: Numerical Recipes in C, 2nd Edition, chapter 2.6, 11.2,		//
//	& 11.3																		//
//																				//
//	candidateA	Pointer to the output location of the A image					//
//	candidateB	Pointer to the output location of the B image					//
//	oldA		Pointer to the A image computed in the previous iteration		//
//	oldB		Pointer to the B image computed in the previous iteration		//
//	width		The width, in pixels, of the image to be compressed				//
//	err			Pointer to memory where errors are reported						//
//																				//
//////////////////////////////////////////////////////////////////////////////////
__device__ void get_optimization_window(const int threadx, const int thready, 
                                        const int width, const int height,
                                        float red[SVD_MAT_HEIGHT],
                                        float green[SVD_MAT_HEIGHT],
                                        float blue[SVD_MAT_HEIGHT],
                                        float A[SVD_MAT_HEIGHT][SVD_MAT_WIDTH]) {
    int i, j, index, pixelx, pixely;
	float dist, modbit, r, gr, b;
	for (j = 0; j < SVD_DIM_Y; j++) {
		for (i = 0; i < SVD_DIM_X; i++) {
			// j is the index of the pixel we want
			index = j*SVD_DIM_X + i;
			pixelx = SVD_OFFSET_X*threadx-1 + i;
			pixely = SVD_OFFSET_Y*thready-1 + j;
            
			// fetch all the necessary values
#ifdef TWO_BPP
//            if (((pixely & 1) == 0 && (pixelx & 1) == 0) ||
//                ((pixely & 1) == 1 && (pixelx & 1) == 1)) {
//                modbit = tex2D(modRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
//            } else {
//                // We are in one of the pixels that is not directly encoded
//                float top, bottom, left, right;
//                top = tex2D(modRef, CLAMP(pixelx, 0, width), CLAMP(pixely-1, 0, height));
//                bottom = tex2D(modRef, CLAMP(pixelx, 0, width), CLAMP(pixely+1, 0, height));
//                left = tex2D(modRef, CLAMP(pixelx-1, 0, width), CLAMP(pixely, 0, height));
//                right = tex2D(modRef, CLAMP(pixelx+1, 0, width), CLAMP(pixely, 0, height));
//                modbit = (top + bottom + right + left) * 0.25f;
//            }
            modbit = tex2D(modRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
#else
			modbit = tex2D(modRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
#endif
			r = tex2D(redCurrentRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
			gr = tex2D(greenCurrentRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
			b = tex2D(blueCurrentRef, CLAMP(pixelx, 0, width), CLAMP(pixely, 0, height));
#ifdef TWO_BY_TWO
            // top left
			dist = tex1Dfetch(svdMatRefTL, index);
			A[index][0] = dist*(1.0f - modbit);
			A[index][1] = dist*modbit;
			red[index] = dist * r;
			green[index] = dist * gr;
			blue[index] = dist * b;
            
            //top right
			dist = tex1Dfetch(svdMatRefTR, index);
			A[index][2] = dist*(1.0f - modbit);
			A[index][3] = dist*modbit;
			red[index] += dist * r;
			green[index] += dist * gr;
			blue[index] += dist * b;
            
            // bottom left
			dist = tex1Dfetch(svdMatRefBL, index);
			A[index][4] = dist*(1.0f - modbit);
			A[index][5] = dist*modbit;
			red[index] += dist * r;
			green[index] += dist * gr;
			blue[index] += dist * b;
            
            // bottom right
			dist = tex1Dfetch(svdMatRefBR, index);
			A[index][6] = dist*(1.0f - modbit);
			A[index][7] = dist*modbit;
			red[index] += dist * r;
			green[index] += dist * gr;
			blue[index] += dist * b;
#else
            // there can be ONLY one!!!
			dist = tex1Dfetch(svdMatRef, index);
			A[index][0] = dist*(1.0f - modbit);
			A[index][1] = dist*modbit;
			red[index] = dist * r;
			green[index] = dist * gr;
			blue[index] = dist * b;
#endif	// TWO_BY_TWO
		}
	}
    
}

#ifdef USE_CHOLESKY
__global__ void cholesky_optimize(unsigned int *candidateA, unsigned int *candidateB,
                                  unsigned int *oldA, unsigned int *oldB, 
                                  int width, int height, int *err) {
    int thready = blockDim.y * blockIdx.y + threadIdx.y;
	int threadx = blockDim.x * blockIdx.x + threadIdx.x;
#ifdef TWO_BY_TWO
	threadx *= 2;
	thready *= 2;
#endif // TWO_BY_TWO
    float A[SVD_MAT_HEIGHT][SVD_MAT_WIDTH];
    float red[SVD_MAT_HEIGHT], green[SVD_MAT_HEIGHT], blue[SVD_MAT_HEIGHT];
    float AtA[SVD_MAT_WIDTH][SVD_MAT_WIDTH];
    float ry[SVD_MAT_WIDTH], gy[SVD_MAT_WIDTH], by[SVD_MAT_WIDTH];
    float rz[SVD_MAT_WIDTH], gz[SVD_MAT_WIDTH], bz[SVD_MAT_WIDTH];
    float rx[SVD_MAT_WIDTH], gx[SVD_MAT_WIDTH], bx[SVD_MAT_WIDTH];
    
    get_optimization_window(threadx, thready, width, height, red, green, blue, A);
    
    // Calculate transpose(A) * A
    int i, j, k;
    for (i = 0; i < SVD_MAT_WIDTH; i++) {
        for (j = 0; j <= i; j++) {
            AtA[j][i] = 0.0f;
            for (k = 0; k < SVD_MAT_HEIGHT; k++) {
                AtA[j][i] += A[k][i] * A[k][j];
            }
        }
    }
    
    // Cholesky decomposition of transpose(A)*A
    float diag[SVD_MAT_WIDTH];
    for (int i = 0; i < SVD_MAT_WIDTH; i++) {
        for (int j = i; j < SVD_MAT_WIDTH; j++) {
            float s = AtA[i][j];
            for (int k = i-1; k >= 0; k--) {
                s -= AtA[i][k] * AtA[j][k];
            }
            if (i == j) {
                diag[i] = sqrtf(s);
            } else {
                AtA[j][i] = s / diag[i];
            }
        }
    }
    for (i = 0; i < SVD_MAT_WIDTH; i++) {
        AtA[i][i] = diag[i];
    }
    
    // Calculate z = transpose(A) * b
    for (i = 0; i < SVD_MAT_WIDTH; i++) {
        rz[i] = 0.0f;
        gz[i] = 0.0f;
        bz[i] = 0.0f;
        for (j = 0; j < SVD_MAT_HEIGHT; j++) {
            rz[i] += A[j][i] * red[j];
            gz[i] += A[j][i] * green[j];
            bz[i] += A[j][i] * blue[j];
        }
    }
    
    // Solve Ly = z
	for (i = 0; i<SVD_MAT_WIDTH; i++) {
		float sumRed = rz[i];
        float sumGreen = gz[i];
        float sumBlue = bz[i];
		for (int k=i-1; k>=0; k--) {
            sumRed -= AtA[i][k] * ry[k];
            sumGreen -= AtA[i][k] * gy[k];
            sumBlue -= AtA[i][k] * by[k];
        }
        ry[i] = sumRed / AtA[i][i];
        gy[i] = sumGreen / AtA[i][i];
        by[i] = sumBlue / AtA[i][i];
	}
    
	// Solve Ltx = y
	for (i = SVD_MAT_WIDTH-1; i>=0; i--) {
		float sumRed = ry[i];
        float sumGreen = gy[i];
        float sumBlue = by[i];
		for (int k=i+1; k < SVD_MAT_WIDTH; k++) {
            sumRed -= AtA[k][i] * rx[k];
            sumGreen -= AtA[k][i] * gx[k];
            sumBlue -= AtA[k][i] * bx[k];
        }
		rx[i] = sumRed / AtA[i][i];
        gx[i] = sumGreen / AtA[i][i];
        bx[i] = sumBlue / AtA[i][i];
	}
#ifdef USE_PIXEL_UPDATE
	// Calculate new representative colors by using a "fix the error" approach.
	// We multiply the pseudo-inverse weight matrix by the difference between the
	// current and original images to get an "update" that is applied to each
	// representative value. We also limit how big the update can be to avoid
	// colors flying out of bounds.
	unsigned int oldColor;
	// Do top left for 2x2 case or only one for 1x1 case
	int tIdx = (width * thready) + threadx;
    int outRed, outGreen, outBlue, repIdx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)rx[0], -16, 16);
	outGreen = CLAMP((int)gx[0], -16, 16);
	outBlue = CLAMP((int)bx[0], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)rx[1], -16, 16);
	outGreen = CLAMP((int)bx[1], -16, 16);
	outBlue = CLAMP((int)gx[1], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#ifdef TWO_BY_TWO
	// Top right case for 2x2
	tIdx = (width * thready) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)rx[2], -16, 16);
	outGreen = CLAMP((int)gx[2], -16, 16);
	outBlue = CLAMP((int)bx[2], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)rx[3], -16, 16);
	outGreen = CLAMP((int)gx[3], -16, 16);
	outBlue = CLAMP((int)bx[3], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
    
	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)rx[4], -16, 16);
	outGreen = CLAMP((int)bx[4], -16, 16);
	outBlue = CLAMP((int)gx[4], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)rx[5], -16, 16);
	outGreen = CLAMP((int)gx[5], -16, 16);
	outBlue = CLAMP((int)bx[5], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
    
	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)rx[6], -16, 16);
	outGreen = CLAMP((int)gx[6], -16, 16);
	outBlue = CLAMP((int)bx[6], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)rx[7], -16, 16);
	outGreen = CLAMP((int)gx[7], -16, 16);
	outBlue = CLAMP((int)bx[7], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#endif // TWO_BY_TWO
#else
	// WARNING - THIS DOES NOT WORK
	// Calculate new representative colors by computing completely new colors.
	// We multiply the pseudo-inverse weight matrix by the actual colors of 
	// the original pixels in our window to get new representative colors.
	// do A candidate. If the new computed color is out of bounds, we discard
	// it and use the old one.
	// Top left case
    tIdx = (width * thready) + threadx;
	if ((frA[0] > 255) || (fgA[0] > 255) || (fbA[0] > 255) || (frA[0] < 0) || (fgA[0] < 0) || (fbA[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[0]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[0] > 255) || (fgB[0] > 255) || (fbB[0] > 255) || (frB[0] < 0) || (fgB[0] < 0) || (fbB[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[0]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#ifdef TWO_BY_TWO
	// Top right case
	tIdx = (width * (thready)) + threadx + 1;
	if ((frA[1] > 255) || (fgA[1] > 255) || (fbA[1] > 255) || (frA[1] < 0) || (fgA[1] < 0) || (fbA[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[1]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[1] > 255) || (fgB[1] > 255) || (fbB[1] > 255) || (frB[1] < 0) || (fgB[1] < 0) || (fbB[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[1]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	if ((frA[2] > 255) || (fgA[2] > 255) || (fbA[2] > 255) || (frA[2] < 0) || (fgA[2] < 0) || (fbA[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[2]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[2] > 255) || (fgB[2] > 255) || (fbB[2] > 255) || (frB[2] < 0) || (fgB[2] < 0) || (fbB[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[2]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	if ((frA[3] > 255) || (fgA[3] > 255) || (fbA[3] > 255) || (frA[3] < 0) || (fgA[3] < 0) || (fbA[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[3]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[3] > 255) || (fgB[3] > 255) || (fbB[3] > 255) || (frB[3] < 0) || (fgB[3] < 0) || (fbB[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[3]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#endif // TWO_BY_TWO
#endif // USE_PIXEL_UPDATE
    
}
#endif // USE_CHOLESKY

#ifdef USE_JAMA_SVD
__global__ void svd_optimize(unsigned int *candidateA, unsigned int *candidateB, 
                             unsigned int *oldA, unsigned int *oldB, 
                             int width, int height, int *err) {
    int thready = blockDim.y * blockIdx.y + threadIdx.y;
	int threadx = blockDim.x * blockIdx.x + threadIdx.x;
#ifdef TWO_BY_TWO
	threadx *= 2;
	thready *= 2;
#endif // TWO_BY_TWO
	const int maxits=30;
	int i, j, k, tIdx = (width*thready) + threadx;
    float w;
	float red[SVD_MAT_HEIGHT],green[SVD_MAT_HEIGHT],
    blue[SVD_MAT_HEIGHT];
    
	// column major order for SVD
	float A[SVD_MAT_HEIGHT][SVD_MAT_WIDTH], s[SVD_MAT_WIDTH], 
    V[SVD_MAT_WIDTH][SVD_MAT_WIDTH], U[SVD_MAT_HEIGHT][SVD_MAT_WIDTH],
    InverseA[SVD_MAT_WIDTH][SVD_MAT_HEIGHT];
    
	// Fetch the weight matrix for the optimization window of the current
	// pair of representative pixels
	get_optimization_window(threadx, thready, width, height, red, green, blue, A);
    
    // Derived from LINPACK code.
    // Initialize.
    const int m = SVD_MAT_HEIGHT;
    const int n = SVD_MAT_WIDTH;
    
    /* Apparently the failing cases are only a proper subset of (m<n), 
	 so let's not throw error.  Correct fix to come later?
     if (m<n) {
     throw new IllegalArgumentException("Jama SVD only works for m >= n"); }
     */
    const int nu = SVD_MAT_WIDTH;
    float e[SVD_MAT_WIDTH], work[SVD_MAT_HEIGHT];
    bool wantu = true;
    bool wantv = true;
    
    // Reduce A to bidiagonal form, storing the diagonal elements
    // in s and the super-diagonal elements in e.
    
    int nct = min(m-1,n);
    int nrt = max(0,min(n-2,m));
    for (int k = 0; k < max(nct,nrt); k++) {
        if (k < nct) {
            
            // Compute the transformation for the k-th column and
            // place the k-th diagonal in s[k].
            // Compute 2-norm of k-th column without under/overflow.
            s[k] = 0;
            for (int i = k; i < m; i++) {
                s[k] = pythag(s[k],A[i][k]);
            }
            if (s[k] != 0.0) {
                if (A[k][k] < 0.0) {
                    s[k] = -s[k];
                }
                for (int i = k; i < m; i++) {
                    A[i][k] /= s[k];
                }
                A[k][k] += 1.0;
            }
            s[k] = -s[k];
        }
        for (int j = k+1; j < n; j++) {
            if ((k < nct) & (s[k] != 0.0))  {
                
                // Apply the transformation.
                
                float t = 0;
                for (int i = k; i < m; i++) {
                    t += A[i][k]*A[i][j];
                }
                t = -t/A[k][k];
                for (int i = k; i < m; i++) {
                    A[i][j] += t*A[i][k];
                }
            }
            
            // Place the k-th row of A into e for the
            // subsequent calculation of the row transformation.
            
            e[j] = A[k][j];
        }
        if (wantu & (k < nct)) {
            
            // Place the transformation in U for subsequent back
            // multiplication.
            
            for (int i = k; i < m; i++) {
                U[i][k] = A[i][k];
            }
        }
        if (k < nrt) {
            
            // Compute the k-th row transformation and place the
            // k-th super-diagonal in e[k].
            // Compute 2-norm without under/overflow.
            e[k] = 0;
            for (int i = k+1; i < n; i++) {
                e[k] = pythag(e[k],e[i]);
            }
            if (e[k] != 0.0) {
                if (e[k+1] < 0.0) {
                    e[k] = -e[k];
                }
                for (int i = k+1; i < n; i++) {
                    e[i] /= e[k];
                }
                e[k+1] += 1.0;
            }
            e[k] = -e[k];
            if ((k+1 < m) & (e[k] != 0.0)) {
                
                // Apply the transformation.
                
                for (int i = k+1; i < m; i++) {
                    work[i] = 0.0;
                }
                for (int j = k+1; j < n; j++) {
                    for (int i = k+1; i < m; i++) {
                        work[i] += e[j]*A[i][j];
                    }
                }
                for (int j = k+1; j < n; j++) {
                    float t = -e[j]/e[k+1];
                    for (int i = k+1; i < m; i++) {
                        A[i][j] += t*work[i];
                    }
                }
            }
            if (wantv) {
                
                // Place the transformation in V for subsequent
                // back multiplication.
                
                for (int i = k+1; i < n; i++) {
                    V[i][k] = e[i];
                }
            }
        }
    }
    
    // Set up the final bidiagonal matrix or order p.
    
    int p = min(n,m+1);
    if (nct < n) {
        s[nct] = A[nct][nct];
    }
    if (m < p) {
        s[p-1] = 0.0;
    }
    if (nrt+1 < p) {
        e[nrt] = A[nrt][p-1];
    }
    e[p-1] = 0.0;
    
    // If required, generate U.
    
    if (wantu) {
        for (int j = nct; j < nu; j++) {
            for (int i = 0; i < m; i++) {
                U[i][j] = 0.0;
            }
            U[j][j] = 1.0;
        }
        for (int k = nct-1; k >= 0; k--) {
            if (s[k] != 0.0) {
                for (int j = k+1; j < nu; j++) {
                    float t = 0;
                    for (int i = k; i < m; i++) {
                        t += U[i][k]*U[i][j];
                    }
                    t = -t/U[k][k];
                    for (int i = k; i < m; i++) {
                        U[i][j] += t*U[i][k];
                    }
                }
                for (int i = k; i < m; i++ ) {
                    U[i][k] = -U[i][k];
                }
                U[k][k] = 1.0 + U[k][k];
                for (int i = 0; i < k-1; i++) {
                    U[i][k] = 0.0;
                }
            } else {
                for (int i = 0; i < m; i++) {
                    U[i][k] = 0.0;
                }
                U[k][k] = 1.0;
            }
        }
    }
    
    // If required, generate V.
    
    if (wantv) {
        for (int k = n-1; k >= 0; k--) {
            if ((k < nrt) & (e[k] != 0.0)) {
                for (int j = k+1; j < nu; j++) {
                    float t = 0;
                    for (int i = k+1; i < n; i++) {
                        t += V[i][k]*V[i][j];
                    }
                    t = -t/V[k+1][k];
                    for (int i = k+1; i < n; i++) {
                        V[i][j] += t*V[i][k];
                    }
                }
            }
            for (int i = 0; i < n; i++) {
                V[i][k] = 0.0;
            }
            V[k][k] = 1.0;
        }
    }
    
    // Main iteration loop for the singular values.
    
    int pp = p-1;
    int iter = 0;
    float eps = pow(2.0,-52.0);
    float tiny = pow(2.0,-966.0);
    while (p > 0) {
        int k,kase;
        
        // Here is where a test for too many iterations would go.
        if (iter > maxits) {
            *err = 1;
            break;
        }
        // This section of the program inspects for
        // negligible elements in the s and e arrays.  On
        // completion the variables kase and k are set as follows.
        
        // kase = 1     if s(p) and e[k-1] are negligible and k<p
        // kase = 2     if s(k) is negligible and k<p
        // kase = 3     if e[k-1] is negligible, k<p, and
        //              s(k), ..., s(p) are not negligible (qr step).
        // kase = 4     if e(p-1) is negligible (convergence).
        
        for (k = p-2; k >= -1; k--) {
            if (k == -1) {
                break;
            }
            if (fabsf(e[k]) <=
                tiny + eps*(fabsf(s[k]) + fabsf(s[k+1]))) {
                e[k] = 0.0;
                break;
            }
        }
        if (k == p-2) {
            kase = 4;
        } else {
            int ks;
            for (ks = p-1; ks >= k; ks--) {
                if (ks == k) {
                    break;
                }
                float t = (ks != p ? fabsf(e[ks]) : 0.) + 
                (ks != k+1 ? fabsf(e[ks-1]) : 0.);
                if (fabsf(s[ks]) <= tiny + eps*t)  {
                    s[ks] = 0.0;
                    break;
                }
            }
            if (ks == k) {
                kase = 3;
            } else if (ks == p-1) {
                kase = 1;
            } else {
                kase = 2;
                k = ks;
            }
        }
        k++;
        
        // Perform the task indicated by kase.
        
        switch (kase) {
                
                // Deflate negligible s(p).
                
            case 1: {
                float f = e[p-2];
                e[p-2] = 0.0;
                for (int j = p-2; j >= k; j--) {
                    float t = pythag(s[j],f);
                    float cs = s[j]/t;
                    float sn = f/t;
                    s[j] = t;
                    if (j != k) {
                        f = -sn*e[j-1];
                        e[j-1] = cs*e[j-1];
                    }
                    if (wantv) {
                        for (int i = 0; i < n; i++) {
                            t = cs*V[i][j] + sn*V[i][p-1];
                            V[i][p-1] = -sn*V[i][j] + cs*V[i][p-1];
                            V[i][j] = t;
                        }
                    }
                }
            }
                break;
                
                // Split at negligible s(k).
                
            case 2: {
                float f = e[k-1];
                e[k-1] = 0.0;
                for (int j = k; j < p; j++) {
                    float t = pythag(s[j],f);
                    float cs = s[j]/t;
                    float sn = f/t;
                    s[j] = t;
                    f = -sn*e[j];
                    e[j] = cs*e[j];
                    if (wantu) {
                        for (int i = 0; i < m; i++) {
                            t = cs*U[i][j] + sn*U[i][k-1];
                            U[i][k-1] = -sn*U[i][j] + cs*U[i][k-1];
                            U[i][j] = t;
                        }
                    }
                }
            }
                break;
                
                // Perform one qr step.
                
            case 3: {
                
                // Calculate the shift.
                
                float scale = max(max(max(max(fabsf(s[p-1]),fabsf(s[p-2])),fabsf(e[p-2])), 
                                                fabsf(s[k])),fabsf(e[k]));
                float sp = s[p-1]/scale;
                float spm1 = s[p-2]/scale;
                float epm1 = e[p-2]/scale;
                float sk = s[k]/scale;
                float ek = e[k]/scale;
                float b = ((spm1 + sp)*(spm1 - sp) + epm1*epm1)/2.0;
                float c = (sp*epm1)*(sp*epm1);
                float shift = 0.0;
                if ((b != 0.0) | (c != 0.0)) {
                    shift = sqrtf(b*b + c);
                    if (b < 0.0) {
                        shift = -shift;
                    }
                    shift = c/(b + shift);
                }
                float f = (sk + sp)*(sk - sp) + shift;
                float g = sk*ek;
                
                // Chase zeros.
                
                for (int j = k; j < p-1; j++) {
                    float t = pythag(f,g);
                    float cs = f/t;
                    float sn = g/t;
                    if (j != k) {
                        e[j-1] = t;
                    }
                    f = cs*s[j] + sn*e[j];
                    e[j] = cs*e[j] - sn*s[j];
                    g = sn*s[j+1];
                    s[j+1] = cs*s[j+1];
                    if (wantv) {
                        for (int i = 0; i < n; i++) {
                            t = cs*V[i][j] + sn*V[i][j+1];
                            V[i][j+1] = -sn*V[i][j] + cs*V[i][j+1];
                            V[i][j] = t;
                        }
                    }
                    t = pythag(f,g);
                    cs = f/t;
                    sn = g/t;
                    s[j] = t;
                    f = cs*e[j] + sn*s[j+1];
                    s[j+1] = -sn*e[j] + cs*s[j+1];
                    g = sn*e[j+1];
                    e[j+1] = cs*e[j+1];
                    if (wantu && (j < m-1)) {
                        for (int i = 0; i < m; i++) {
                            t = cs*U[i][j] + sn*U[i][j+1];
                            U[i][j+1] = -sn*U[i][j] + cs*U[i][j+1];
                            U[i][j] = t;
                        }
                    }
                }
                e[p-2] = f;
                iter = iter + 1;
            }
                break;
                
                // Convergence.
                
            case 4: {
                
                // Make the singular values positive.
                
                if (s[k] <= 0.0) {
                    s[k] = (s[k] < 0.0 ? -s[k] : 0.0);
                    if (wantv) {
                        for (int i = 0; i <= pp; i++) {
                            V[i][k] = -V[i][k];
                        }
                    }
                }
                
                // Order the singular values.
                
//                while (k < pp) {
//                    if (s[k] >= s[k+1]) {
//                        break;
//                    }
//                    float t = s[k];
//                    s[k] = s[k+1];
//                    s[k+1] = t;
//                    if (wantv && (k < n-1)) {
//                        for (int i = 0; i < n; i++) {
//                            t = V[i][k+1]; V[i][k+1] = V[i][k]; V[i][k] = t;
//                        }
//                    }
//                    if (wantu && (k < m-1)) {
//                        for (int i = 0; i < m; i++) {
//                            t = U[i][k+1]; U[i][k+1] = U[i][k]; U[i][k] = t;
//                        }
//                    }
//                    k++;
//                }  
                iter = 0;
                p--;
            }
                break;
        }
    }
    
    // begin constructing inverse matrix
	// inverse(A) = V * inverse(W) * transpose(U)
    
	// Step 1: transpose(A) = inverse(W) * transpose(U)
	for (i = 0; i < SVD_MAT_WIDTH; i++) {
		w = 1.0f / s[i];
		for (j=0; j < SVD_MAT_HEIGHT; j++) {
			U[j][i] *= w;
		}
	}
    
	// Step 2: inverse(A) = V * transpose(U)
	for (i = 0; i < SVD_MAT_WIDTH; i++) {			// rows of Eigenbasis
		for (j = 0; j < SVD_MAT_HEIGHT; j++) {		// colums of U transpose
			InverseA[i][j] = 0.0f;
			for (k = 0; k < SVD_MAT_WIDTH; k++) {	// dimension of overlap
				// U is stored as a transpose so we must flip indices accordingly
				// In other words U[row][column] becomes U[column][row]
				InverseA[i][j] += V[i][k] * U[j][k];
			}
		}
	}
    
	// compute optimized A and B values
	float frA[4], fgA[4], fbA[4], frB[4], fgB[4], fbB[4];
	int outRed, outGreen, outBlue, repIdx;
	for (j = 0; j < SVD_MAT_WIDTH; j+=2) {
		repIdx = (j>>1);	// repIdx = j / 2
		frA[repIdx] = fgA[repIdx] = fbA[repIdx] = frB[repIdx] = fgB[repIdx] = fbB[repIdx] = 0.0f;
		for (i = 0; i < SVD_MAT_HEIGHT; i++) {
			frA[repIdx] += InverseA[j][i] * red[i];
			fgA[repIdx] += InverseA[j][i] * green[i];
			fbA[repIdx] += InverseA[j][i] * blue[i];
			frB[repIdx] += InverseA[j+1][i] * red[i];
			fgB[repIdx] += InverseA[j+1][i] * green[i];
			fbB[repIdx] += InverseA[j+1][i] * blue[i];
		}
	}
    
#ifdef USE_PIXEL_UPDATE
	// Calculate new representative colors by using a "fix the error" approach.
	// We multiply the pseudo-inverse weight matrix by the difference between the
	// current and original images to get an "update" that is applied to each
	// representative value. We also limit how big the update can be to avoid
	// colors flying out of bounds.
	unsigned int oldColor;
	// Do top left for 2x2 case or only one for 1x1 case
	tIdx = (width * thready) + threadx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[0], -16, 16);
	outGreen = CLAMP((int)fgA[0], -16, 16);
	outBlue = CLAMP((int)fbA[0], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[0], -16, 16);
	outGreen = CLAMP((int)fgB[0], -16, 16);
	outBlue = CLAMP((int)fbB[0], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#ifdef TWO_BY_TWO
	// Top right case for 2x2
	tIdx = (width * thready) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[1], -16, 16);
	outGreen = CLAMP((int)fgA[1], -16, 16);
	outBlue = CLAMP((int)fbA[1], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[1], -16, 16);
	outGreen = CLAMP((int)fgB[1], -16, 16);
	outBlue = CLAMP((int)fbB[1], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
    
	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[2], -16, 16);
	outGreen = CLAMP((int)fgA[2], -16, 16);
	outBlue = CLAMP((int)fbA[2], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[2], -16, 16);
	outGreen = CLAMP((int)fgB[2], -16, 16);
	outBlue = CLAMP((int)fbB[2], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
    
	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[3], -16, 16);
	outGreen = CLAMP((int)fgA[3], -16, 16);
	outBlue = CLAMP((int)fbA[3], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];
    
	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[3], -16, 16);
	outGreen = CLAMP((int)fgB[3], -16, 16);
	outBlue = CLAMP((int)fbB[3], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
                                 CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
                                 CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
                                 CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#endif // TWO_BY_TWO
#else
	// WARNING - THIS DOES NOT WORK
	// Calculate new representative colors by computing completely new colors.
	// We multiply the pseudo-inverse weight matrix by the actual colors of 
	// the original pixels in our window to get new representative colors.
	// do A candidate. If the new computed color is out of bounds, we discard
	// it and use the old one.
	// Top left case
    tIdx = (width * thready) + threadx;
	if ((frA[0] > 255) || (fgA[0] > 255) || (fbA[0] > 255) || (frA[0] < 0) || (fgA[0] < 0) || (fbA[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[0]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[0] > 255) || (fgB[0] > 255) || (fbB[0] > 255) || (frB[0] < 0) || (fgB[0] < 0) || (fbB[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[0]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#ifdef TWO_BY_TWO
	// Top right case
	tIdx = (width * (thready)) + threadx + 1;
	if ((frA[1] > 255) || (fgA[1] > 255) || (fbA[1] > 255) || (frA[1] < 0) || (fgA[1] < 0) || (fbA[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[1]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[1] > 255) || (fgB[1] > 255) || (fbB[1] > 255) || (frB[1] < 0) || (fgB[1] < 0) || (fbB[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[1]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	if ((frA[2] > 255) || (fgA[2] > 255) || (fbA[2] > 255) || (frA[2] < 0) || (fgA[2] < 0) || (fbA[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[2]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[2] > 255) || (fgB[2] > 255) || (fbB[2] > 255) || (frB[2] < 0) || (fgB[2] < 0) || (fbB[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[2]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	if ((frA[3] > 255) || (fgA[3] > 255) || (fbA[3] > 255) || (frA[3] < 0) || (fgA[3] < 0) || (fbA[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[3]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
    
	// do B candidate
	if ((frB[3] > 255) || (fgB[3] > 255) || (fbB[3] > 255) || (frB[3] < 0) || (fgB[3] < 0) || (fbB[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[3]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#endif // TWO_BY_TWO
#endif // USE_PIXEL_UPDATE
}
#endif // USE_JAMA_SVD

#ifdef USE_SVD
__global__ void svd_optimize(unsigned int *candidateA, unsigned int *candidateB,
									unsigned int *oldA, unsigned int *oldB, 
									int width,int height,  int *err) {
	int thready = blockDim.y * blockIdx.y + threadIdx.y;
	int threadx = blockDim.x * blockIdx.x + threadIdx.x;
#ifdef TWO_BY_TWO
	threadx *= 2;
	thready *= 2;
#endif // TWO_BY_TWO
	const int maxits=30;
	int tIdx = (width*thready) + threadx;
	int i,its,j,jj,k,l=0,nm=0;
	float red[SVD_MAT_HEIGHT],green[SVD_MAT_HEIGHT],
		blue[SVD_MAT_HEIGHT];
	bool flag;
	float anorm,c,f,g,h,s,scale,w,x,y,z,rv[SVD_MAT_WIDTH]; 
	// column major order for SVD
	float A[SVD_MAT_HEIGHT][SVD_MAT_WIDTH], Eigenvalues[SVD_MAT_WIDTH], 
    Eigenbasis[SVD_MAT_WIDTH][SVD_MAT_WIDTH]; 
		//InverseA[SVD_MAT_WIDTH][SVD_MAT_HEIGHT];

	// Fetch the weight matrix for the optimization window of the current
	// pair of representative pixels
	get_optimization_window(threadx, thready, width, height, red, green, blue, A);

	// Householder reduction to bidiagonal form
	g = scale = anorm = 0.0f;
	for (i=0; i<SVD_MAT_WIDTH; i++) {
		l = i+1;
		rv[i] = scale*g;
		g = s = scale = 0.0f;
		if (i<SVD_MAT_HEIGHT) {
			for (k=i; k<SVD_MAT_HEIGHT; k++) scale+= fabsf(A[k][i]);
			if (scale!=0.0f) {
				for (k=i; k<SVD_MAT_HEIGHT; k++) {
					A[k][i]/= scale;
					s+= SQR(A[k][i]);
				}
				f = A[i][i];
				g = -sign(sqrtf(s),f);
				h = f*g-s;
				A[i][i] = f-g;
				for (j=l; j<SVD_MAT_WIDTH; j++) {
				  for (s=0.0f,k=i; k<SVD_MAT_HEIGHT; k++) s+= A[k][i]*A[k][j];
					f = s/h;
					for (k=i; k<SVD_MAT_HEIGHT; k++) A[k][j]+= f*A[k][i];
				}
				for (k=i; k<SVD_MAT_HEIGHT; k++) A[k][i]*= scale;
			}
		}
		Eigenvalues[i] = scale*g;
		g = s = scale = 0.0f;
		if (i<SVD_MAT_HEIGHT && i!=SVD_MAT_WIDTH-1) {
			for (k=l; k<SVD_MAT_WIDTH; k++) scale+= fabsf(A[i][k]);
			if (scale!=0.0f)  {
				for(k=l; k<SVD_MAT_WIDTH; k++) {
					A[i][k]/= scale;
					s+= SQR(A[i][k]);
				}
				f = A[i][l];
				g = -sign(sqrtf(s),f);
				h = f*g-s;
				A[i][l] = f-g;
				for (k=l; k<SVD_MAT_WIDTH; k++) rv[k] = A[i][k]/h;
				for (j=l; j<SVD_MAT_HEIGHT; j++) {
				  for(s=0.0f,k=l; k<SVD_MAT_WIDTH; k++) s+= A[j][k]*A[i][k];
					for(k=l; k<SVD_MAT_WIDTH; k++) A[j][k]+= s*rv[k];
				}
				for(k=l; k<SVD_MAT_WIDTH; k++) A[i][k]*= scale;
			}
		}
		anorm = fmaxf(anorm,(fabsf(Eigenvalues[i])+fabsf(rv[i])));
	}
	
	// Accumulate right-hand side updates
	for(i=SVD_MAT_WIDTH-1; i>=0; i--) {
		if (i<SVD_MAT_WIDTH-1) {
		  if (g!=0.0f) {
				for (j=l; j<SVD_MAT_WIDTH; j++) Eigenbasis[j][i] = (A[i][j]/A[i][l])/g;
				for (j=l; j<SVD_MAT_WIDTH; j++) {
				  for(s=0.0f,k=l; k<SVD_MAT_WIDTH; k++) s+= A[i][k]*Eigenbasis[k][j];
					for(k=l; k<SVD_MAT_WIDTH; k++) Eigenbasis[k][j]+= s*Eigenbasis[k][i];
				}
			}
			for (j=l; j<SVD_MAT_WIDTH; j++) Eigenbasis[i][j] = Eigenbasis[j][i] = 0.0f;
		}
		Eigenbasis[i][i] = 1.0f;
		g = rv[i];
		l = i;
	}
		
	// Accumulate left-hand side updates
	for (i=min(SVD_MAT_HEIGHT-1,SVD_MAT_WIDTH-1); i>=0; i--) {
		l = i+1;
		g = Eigenvalues[i];
		for(j=l;j<SVD_MAT_WIDTH;j++) A[i][j] = 0.0f;
		if (g!=0.0f) {
		  g = 1.0f/g;
			for (j=l; j<SVD_MAT_WIDTH; j++) {
			  for (s=0.0f,k=l; k<SVD_MAT_HEIGHT; k++) s+= A[k][i]*A[k][j];
				f = (s/A[i][i])*g;
				for (k=i; k<SVD_MAT_HEIGHT; k++) A[k][j]+= f*A[k][i];
			}
			for (j=i; j<SVD_MAT_HEIGHT; j++) A[j][i]*= g;
		}
		else for (j=i; j<SVD_MAT_HEIGHT; j++) A[j][i] = 0.0f;
		A[i][i]+= 1.0f;
	}
	
	// diagonalization of the bidiagonal form: loop over singular values, and
	// over allowed iterations
	for (k=SVD_MAT_WIDTH-1; k>=0; k--) {
		for (its=0; its<maxits; its++) {
			flag = true;
			for (l=k; l>=0; l--) {			// test for splitting
				nm = l-1;
				if ((float)(fabsf(rv[l])+anorm) == anorm) {
					flag =  false;
					break;
				}
				if ((float)(fabsf(Eigenvalues[nm])+anorm) == anorm) break;
			}
			if (flag) {
			  c = 0.0f;
			  s = 1.0f;
				for (i=l; i<=k; i++) {
					f = s*rv[i];
					rv[i]*= c;
					if ((float)(fabsf(f)+anorm) == anorm) break;
					g = Eigenvalues[i];
					h = pythag(f,g);
					Eigenvalues[i] = h;
					h = 1.0f/h;
					c = g*h;
					s = -f*h;
					for (j=0; j<SVD_MAT_HEIGHT; j++) {
						y = A[j][nm];
						z = A[j][i];
						A[j][nm] = y*c+z*s;
						A[j][i]  = z*c-y*s;
					}
				}
			}
			z = Eigenvalues[k];
			if (l==k) {					// convergence
				if(z<0.0f) {			// singular value is made non-negative
					Eigenvalues[k] = -z;
					for(j=0; j<SVD_MAT_WIDTH; j++) Eigenbasis[j][k] *= -1.0f;
				}
				break;
			}
			
			if(its>=maxits) { *err = 1; } // error check
			x = Eigenvalues[l];			// shift from bottom 2-by-2 minor
			nm = k-1;
			y = Eigenvalues[nm];
			g = rv[nm];
			h = rv[k];
			f = ((y-z)*(y+z)+(g-h)*(g+h))/(2.0f*h*y);
			g = pythag(f,1.0);
			f = ((x-z)*(x+z)+h*((y/(f+sign(g,f)))-h))/x;
			c = s = 1.0f;				// next QR transformation
			for (j=l; j<=nm; j++) {
				i = j+1;
				g = rv[i];
				y = Eigenvalues[i];
				h = s*g;
				g = c*g;
				z = pythag(f,h);
				rv[j] = z;
				c = f/z;
				s = h/z;
				f = x*c+g*s;
				g = g*c-x*s;
				h = y*s;
				y*= c;
				for(jj=0;jj<SVD_MAT_WIDTH;jj++) {
					x = Eigenbasis[jj][j];
					z = Eigenbasis[jj][i];
					Eigenbasis[jj][j] = x*c+z*s;
					Eigenbasis[jj][i] = z*c-x*s;
				}
				z = pythag(f,h);
				Eigenvalues[j] = z;		// rotation can be arbitrary if z = 0
				if (z!=0.0f) {
				  z = 1.0f / z;
					c = f*z;
					s = h*z;
				}
				f = c*g+s*y;
				x = c*y-s*g;
				for (jj=0; jj<SVD_MAT_HEIGHT; jj++) {
					y = A[jj][j];
					z = A[jj][i];
					A[jj][j] = y*c+z*s;
					A[jj][i] = z*c-y*s;
				}
			}
			rv[l] = 0.0f;
			rv[k] = f;
			Eigenvalues[k] = x;
		}
	}

	// begin constructing inverse matrix
	// inverse(A) = V * inverse(W) * transpose(A)

	// Step 1: transpose(A) = inverse(W) * transpose(A)
	for (i = 0; i < SVD_MAT_WIDTH; i++) {
		w = 1.0f / Eigenvalues[i];
		for (j=0; j < SVD_MAT_HEIGHT; j++) {
			A[j][i] *= w;
		}
	}

//	// Step 2: inverse(A) = V * transpose(A)
//	for (i = 0; i < SVD_MAT_WIDTH; i++) {			// rows of Eigenbasis
//		for (j = 0; j < SVD_MAT_HEIGHT; j++) {		// colums of A transpose
//			InverseA[i][j] = 0.0f;
//			for (k = 0; k < SVD_MAT_WIDTH; k++) {	// dimension of overlap
//				// A is stored as a transpose so we must flip indices accordingly
//				// In other words A[row][column] becomes A[column][row]
//				InverseA[i][j] += Eigenbasis[i][k] * A[j][k];
//			}
//		}
//	}
//    
//    // compute optimized A and B values
//	float frA[4], fgA[4], fbA[4], frB[4], fgB[4], fbB[4];
//    float tempR[SVD_MAT_WIDTH], tempG[SVD_MAT_WIDTH], tempB[SVD_MAT_WIDTH];
//	int outRed, outGreen, outBlue, repIdx;
//    for (j = 0; j < SVD_MAT_WIDTH; j+=2) {
//		repIdx = (j>>1);	// repIdx = j / 2
//		frA[repIdx] = fgA[repIdx] = fbA[repIdx] = frB[repIdx] = fgB[repIdx] = fbB[repIdx] = 0.0f;
//		for (i = 0; i < SVD_MAT_HEIGHT; i++) {
//			frA[repIdx] += InverseA[j][i] * red[i];
//			fgA[repIdx] += InverseA[j][i] * green[i];
//			fbA[repIdx] += InverseA[j][i] * blue[i];
//			frB[repIdx] += InverseA[j+1][i] * red[i];
//			fgB[repIdx] += InverseA[j+1][i] * green[i];
//			fbB[repIdx] += InverseA[j+1][i] * blue[i];
//		}
//	}

    // compute optimized A and B values
	float frA[4], fgA[4], fbA[4], frB[4], fgB[4], fbB[4];
    float tempR[SVD_MAT_WIDTH], tempG[SVD_MAT_WIDTH], tempB[SVD_MAT_WIDTH];
	int outRed, outGreen, outBlue, repIdx;
    for (i = 0; i < SVD_MAT_WIDTH; i++) {
        tempR[i] = 0.0f;
        tempG[i] = 0.0f;
        tempB[i] = 0.0f;
        for (k = 0; k < SVD_MAT_HEIGHT; k++) {
            tempR[i] += A[k][i] * red[k];
            tempG[i] += A[k][i] * green[k];
            tempB[i] += A[k][i] * blue[k];
        }
    }
	
    for (j = 0; j < SVD_MAT_WIDTH; j+=2) {
		repIdx = (j>>1);	// repIdx = j / 2
		frA[repIdx] = fgA[repIdx] = fbA[repIdx] = frB[repIdx] = fgB[repIdx] = fbB[repIdx] = 0.0f;
		for (i = 0; i < SVD_MAT_WIDTH; i++) {
			frA[repIdx] += Eigenbasis[j][i] * tempR[i];
			fgA[repIdx] += Eigenbasis[j][i] * tempG[i];
			fbA[repIdx] += Eigenbasis[j][i] * tempB[i];
			frB[repIdx] += Eigenbasis[j+1][i] * tempR[i];
			fgB[repIdx] += Eigenbasis[j+1][i] * tempG[i];
			fbB[repIdx] += Eigenbasis[j+1][i] * tempB[i];
		}
	}

#ifdef USE_PIXEL_UPDATE
	// Calculate new representative colors by using a "fix the error" approach.
	// We multiply the pseudo-inverse weight matrix by the difference between the
	// current and original images to get an "update" that is applied to each
	// representative value. We also limit how big the update can be to avoid
	// colors flying out of bounds.
	unsigned int oldColor;
	// Do top left for 2x2 case or only one for 1x1 case
	tIdx = (width * thready) + threadx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[0], -16, 16);
	outGreen = CLAMP((int)fgA[0], -16, 16);
	outBlue = CLAMP((int)fbA[0], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];

	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[0], -16, 16);
	outGreen = CLAMP((int)fgB[0], -16, 16);
	outBlue = CLAMP((int)fbB[0], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#ifdef TWO_BY_TWO
	// Top right case for 2x2
	tIdx = (width * thready) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[1], -16, 16);
	outGreen = CLAMP((int)fgA[1], -16, 16);
	outBlue = CLAMP((int)fbA[1], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];

	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[1], -16, 16);
	outGreen = CLAMP((int)fgB[1], -16, 16);
	outBlue = CLAMP((int)fbB[1], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];

	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[2], -16, 16);
	outGreen = CLAMP((int)fgA[2], -16, 16);
	outBlue = CLAMP((int)fbA[2], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];

	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[2], -16, 16);
	outGreen = CLAMP((int)fgB[2], -16, 16);
	outBlue = CLAMP((int)fbB[2], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];

	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	oldColor = oldA[tIdx];
	outRed = CLAMP((int)frA[3], -16, 16);
	outGreen = CLAMP((int)fgA[3], -16, 16);
	outBlue = CLAMP((int)fbA[3], -16, 16);
	candidateA[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldA[tIdx] = candidateA[tIdx];

	oldColor = oldB[tIdx];
	outRed = CLAMP((int)frB[3], -16, 16);
	outGreen = CLAMP((int)fgB[3], -16, 16);
	outBlue = CLAMP((int)fbB[3], -16, 16);
	candidateB[tIdx] = MAKE_ARGB(
		CLAMP((int)MAKE_RED_PIXEL(oldColor) + outRed, 0, 255),
		CLAMP((int)MAKE_GREEN_PIXEL(oldColor) + outGreen, 0, 255),
		CLAMP((int)MAKE_BLUE_PIXEL(oldColor) + outBlue, 0, 255));
	oldB[tIdx] = candidateB[tIdx];
#endif // TWO_BY_TWO
#else
	// WARNING - THIS DOES NOT WORK
	// Calculate new representative colors by computing completely new colors.
	// We multiply the pseudo-inverse weight matrix by the actual colors of 
	// the original pixels in our window to get new representative colors.
	// do A candidate. If the new computed color is out of bounds, we discard
	// it and use the old one.
	// Top left case
	if ((frA[0] > 255) || (fgA[0] > 255) || (fbA[0] > 255) || (frA[0] < 0) || (fgA[0] < 0) || (fbA[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[0]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// do B candidate
	if ((frB[0] > 255) || (fgB[0] > 255) || (fbB[0] > 255) || (frB[0] < 0) || (fgB[0] < 0) || (fbB[0] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[0]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[0]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[0]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#ifdef TWO_BY_TWO
	// Top right case
	tIdx = (width * (thready)) + threadx + 1;
	if ((frA[1] > 255) || (fgA[1] > 255) || (fbA[1] > 255) || (frA[1] < 0) || (fgA[1] < 0) || (fbA[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[1]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// do B candidate
	if ((frB[1] > 255) || (fgB[1] > 255) || (fbB[1] > 255) || (frB[1] < 0) || (fgB[1] < 0) || (fbB[1] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[1]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[1]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[1]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// Bottom left case
	tIdx = (width * (thready+1)) + threadx;
	if ((frA[2] > 255) || (fgA[2] > 255) || (fbA[2] > 255) || (frA[2] < 0) || (fgA[2] < 0) || (fbA[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[2]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// do B candidate
	if ((frB[2] > 255) || (fgB[2] > 255) || (fbB[2] > 255) || (frB[2] < 0) || (fgB[2] < 0) || (fbB[2] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[2]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[2]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[2]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// Bottom right case
	tIdx = (width * (thready+1)) + threadx + 1;
	if ((frA[3] > 255) || (fgA[3] > 255) || (fbA[3] > 255) || (frA[3] < 0) || (fgA[3] < 0) || (fbA[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldA[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldA[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldA[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frA[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgA[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbA[3]), 0, 255);
	}
	candidateA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldA[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);

	// do B candidate
	if ((frB[3] > 255) || (fgB[3] > 255) || (fbB[3] > 255) || (frB[3] < 0) || (fgB[3] < 0) || (fbB[3] < 0)) {
		outRed = MAKE_RED_PIXEL(oldB[tIdx]);
		outGreen = MAKE_GREEN_PIXEL(oldB[tIdx]);
		outBlue = MAKE_BLUE_PIXEL(oldB[tIdx]);
	} else {
		outRed = CLAMP(__float2uint_rn(frB[3]), 0, 255);
		outGreen = CLAMP(__float2uint_rn(fgB[3]), 0, 255);
		outBlue = CLAMP(__float2uint_rn(fbB[3]), 0, 255);
	}
	candidateB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
	oldB[tIdx] = MAKE_ARGB(outRed, outGreen, outBlue);
#endif // TWO_BY_TWO
#endif // USE_PIXEL_UPDATE
}
#endif // USE_SVD
//////////////////////////////////////////////////////////////////////////////////
//								  Optimization									//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//								 Modulation Bits								//
//////////////////////////////////////////////////////////////////////////////////
#ifdef TWO_BPP  
__global__ void compute_modulation_mode(unsigned int *a, unsigned int *b,
                                        unsigned int *mode, float *modbits,
                                        int *red, int *green, int *blue, int width) {
    unsigned int thready = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int threadx = blockDim.x * blockIdx.x + threadIdx.x;
    
    unsigned int x = 8 * threadx;
    unsigned int y = 4 * thready;
    
    float mode0_err = 0.0f, mode1_err = 0.0f;
    
    int i, j;
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 8; j++) {
            unsigned int idx = (y+i)*width + (x+j);
            unsigned int apixel = a[idx];
            int a_red = MAKE_RED_PIXEL(apixel);
            int a_green = MAKE_GREEN_PIXEL(apixel);
            int a_blue = MAKE_BLUE_PIXEL(apixel);
            
            unsigned int bpixel = b[idx];
            int b_red = MAKE_RED_PIXEL(bpixel);
            int b_green = MAKE_GREEN_PIXEL(bpixel);
            int b_blue = MAKE_BLUE_PIXEL(bpixel);
            
            unsigned int opixel = tex2D(origRef, x + j, y + i);
            int o_red = MAKE_RED_PIXEL(opixel);
            int o_green = MAKE_GREEN_PIXEL(opixel);
            int o_blue = MAKE_BLUE_PIXEL(opixel);
            
            float mod = modbits[idx];
            float r_mod = 1.0f - mod;
            mode1_err += SQR(o_red - ((float)b_red*mod + (float)a_red*r_mod));
            mode1_err += SQR(o_green - ((float)b_green*mod + (float)a_green*r_mod));
            mode1_err += SQR(o_blue - ((float)b_blue*mod + (float)a_blue*r_mod));
            
            if (mod < 0.5f) {
                mod = 0.0f;
            } else {
                mod = 1.0f;
            }
            r_mod = 1.0f - mod;
            mode0_err += SQR(o_red - ((float)b_red*mod + (float)a_red*r_mod));
            mode0_err += SQR(o_green - ((float)b_green*mod + (float)a_green*r_mod));
            mode0_err += SQR(o_blue - ((float)b_blue*mod + (float)a_blue*r_mod));
        }
    }
    
    if (mode0_err < mode1_err) {
        mode[thready*width + threadx] = 0;
        
        for (i = 0; i < 4; i++) {
            for (j = 0; j < 8; j++) {
                unsigned int idx = (y+i)*width + (x+j);
                if (modbits[idx] < 0.5f) {
                    modbits[idx] = 0.0f;
                } else {
                    modbits[idx] = 1.0f;
                }
                
                unsigned int apixel = a[idx];
                int a_red = MAKE_RED_PIXEL(apixel);
                int a_green = MAKE_GREEN_PIXEL(apixel);
                int a_blue = MAKE_BLUE_PIXEL(apixel);
                
                unsigned int bpixel = b[idx];
                int b_red = MAKE_RED_PIXEL(bpixel);
                int b_green = MAKE_GREEN_PIXEL(bpixel);
                int b_blue = MAKE_BLUE_PIXEL(bpixel);
                
                unsigned int opixel = tex2D(origRef, x + j, y + i);
                int o_red = MAKE_RED_PIXEL(opixel);
                int o_green = MAKE_GREEN_PIXEL(opixel);
                int o_blue = MAKE_BLUE_PIXEL(opixel);
                float mod = modbits[idx];
                float r_mod = 1.0f - mod;
                red[idx] += o_red - ((float)b_red*mod + (float)a_red*r_mod);
                green[idx] += o_green - ((float)b_green*mod + (float)a_green*r_mod);
                blue[idx] += o_blue - ((float)b_blue*mod + (float)a_blue*r_mod);
            }
        }
    } else {
        mode[thready*width + threadx] = 1;
    }
}
#endif // TWO_BPP
// compute modulation bits based on reconstruction of original image
__global__ void compute_modulation_bits(unsigned int *a, unsigned int *b,
										int *redCurrent, int *greenCurrent, 
										int *blueCurrent, float *mod, int width) {
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = (width * y) + x;

	unsigned int apixel = a[idx];
	int a_red = MAKE_RED_PIXEL(apixel);
	int a_green = MAKE_GREEN_PIXEL(apixel);
	int a_blue = MAKE_BLUE_PIXEL(apixel);

	unsigned int bpixel = b[idx];
	int b_red = MAKE_RED_PIXEL(bpixel);
	int b_green = MAKE_GREEN_PIXEL(bpixel);
	int b_blue = MAKE_BLUE_PIXEL(bpixel);

	unsigned int opixel = tex2D(origRef, x, y);
	int o_red = MAKE_RED_PIXEL(opixel);
	int o_green = MAKE_GREEN_PIXEL(opixel);
	int o_blue = MAKE_BLUE_PIXEL(opixel);

	int3 deltaA = make_int3(o_red - a_red, o_green - a_green, o_blue - a_blue);
	int3 deltaB = make_int3(o_red - b_red, o_green - b_green, o_blue - b_blue);
	int3 delta38 = make_int3(o_red - (FIVE_EIGHTHS*a_red + THREE_EIGHTHS*b_red),
		o_green - (FIVE_EIGHTHS*a_green + THREE_EIGHTHS*b_green),
		o_blue - (FIVE_EIGHTHS*a_blue + THREE_EIGHTHS*b_blue));
	int3 delta58 = make_int3(o_red - (THREE_EIGHTHS*a_red + FIVE_EIGHTHS*b_red),
		o_green - (THREE_EIGHTHS*a_green + FIVE_EIGHTHS*b_green),
		o_blue - (THREE_EIGHTHS*a_blue + FIVE_EIGHTHS*b_blue));

	int dotA = SQR(deltaA.x) + SQR(deltaA.y) + SQR(deltaA.z);
	int dotB = SQR(deltaB.x) + SQR(deltaB.y) + SQR(deltaB.z);
	int dot38 = SQR(delta38.x) + SQR(delta38.y) + SQR(delta38.z);
	int dot58 = SQR(delta58.x) + SQR(delta58.y) + SQR(delta58.z);
	int dotMin = min(min(dotA, dotB), min(dot38, dot58));
	float modbit;

	if (dotMin == dotA) {
		modbit = 0.0f;
	} else if (dotMin == dot38) {
		modbit = THREE_EIGHTHS;
	} else if (dotMin == dot58) {
		modbit = FIVE_EIGHTHS;
	} else {
		modbit = 1.0f;
	}
	int3 diff = make_int3(b_red - a_red, b_green - a_green, b_blue - a_blue);

	float dot = (float)(deltaA.x*diff.x + deltaA.y*diff.y + deltaA.z*diff.z) /
				(float)(diff.x*diff.x + diff.y*diff.y + diff.z*diff.z);

	/*if (dot < 0.1875f) {
		modbit = 0.000005f;
	} else if (dot < 0.5f) {
		modbit = THREE_EIGHTHS;
	} else if (dot < 0.8125f) {
		modbit = FIVE_EIGHTHS;
	} else {
		modbit = 0.999995f;
	}*/
#ifdef USE_PIXEL_UPDATE
	float r_modbit = 1.0f - modbit;
	int c_red = ((float)b_red*modbit + (float)a_red*r_modbit);
	int c_green = ((float)b_green*modbit + (float)a_green*r_modbit);
	int c_blue = ((float)b_blue*modbit + (float)a_blue*r_modbit);

	redCurrent[idx] = o_red - c_red;
	greenCurrent[idx] = o_green - c_green;
	blueCurrent[idx] = o_blue - c_blue;
#else
	// WARNING - THIS DOES NOT WORK
	redCurrent[idx] = o_red;
	greenCurrent[idx] = o_green;
	blueCurrent[idx] = o_blue;
#endif
	mod[idx] = modbit;
}
//////////////////////////////////////////////////////////////////////////////////
//								 Modulation Bits								//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//								A and B prototypes								//
//////////////////////////////////////////////////////////////////////////////////
__global__ void make_a_b_prototypes(unsigned int *a_proto, unsigned int *b_proto,
									int width, int height) {
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = (width * y) + x;

	// get original, delta, and axis pixel
	int opixel = tex2D(origRef, x, y);
	int fpixel = tex2D(texRef, x, y);

	int orig_red = MAKE_RED_PIXEL(opixel);
	int orig_green = MAKE_GREEN_PIXEL(opixel);
	int orig_blue = MAKE_BLUE_PIXEL(opixel);
	int delta_red = orig_red - MAKE_RED_PIXEL(fpixel);
	int delta_green = orig_green - MAKE_GREEN_PIXEL(fpixel);
	int delta_blue = orig_blue - MAKE_BLUE_PIXEL(fpixel);
	int axis_red = CLAMP(abs(delta_red), 0, 255);
	int axis_green = CLAMP(abs(delta_green), 0, 255);
	int axis_blue = CLAMP(abs(delta_blue), 0, 255);

	// (delta dot axis) / (axis dot axis)
	int dot = ((delta_red * axis_red) + (delta_green * axis_green) + 
			  (delta_blue * axis_blue)) / ((axis_red * axis_red) + 
			  (axis_green * axis_green) + (axis_blue * axis_blue));

	/*if (dot < 0) {
		a_proto[idx] = MAKE_ARGB((CLAMP(orig_red + axis_red * dot, 0, 255)), 
								 (CLAMP(orig_green + axis_green * dot, 0, 255)), 
								 (CLAMP(orig_blue + axis_blue * dot, 0, 255)));

		b_proto[idx] = MAKE_ARGB((CLAMP(orig_red - axis_red * dot, 0, 255)), 
								 (CLAMP(orig_green - axis_green * dot, 0, 255)), 
								 (CLAMP(orig_blue - axis_blue * dot, 0, 255)));
	} else {
		b_proto[idx] = MAKE_ARGB((CLAMP(orig_red + axis_red * dot, 0, 255)), 
								 (CLAMP(orig_green + axis_green * dot, 0, 255)), 
								 (CLAMP(orig_blue + axis_blue * dot, 0, 255)));

		a_proto[idx] = MAKE_ARGB((CLAMP(orig_red - axis_red * dot, 0, 255)), 
								 (CLAMP(orig_green - axis_green * dot, 0, 255)), 
								 (CLAMP(orig_blue - axis_blue * dot, 0, 255)));
	}*/
	b_proto[idx] = MAKE_ARGB((CLAMP(orig_red + axis_red, 0, 255)), 
		(CLAMP(orig_green + axis_green, 0, 255)), 
		(CLAMP(orig_blue + axis_blue, 0, 255)));

	a_proto[idx] = MAKE_ARGB((CLAMP(orig_red - axis_red, 0, 255)), 
		(CLAMP(orig_green - axis_green, 0, 255)), 
		(CLAMP(orig_blue - axis_blue, 0, 255)));
}
//////////////////////////////////////////////////////////////////////////////////
//								A and B prototypes								//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//								bilinear upscaling								//
//////////////////////////////////////////////////////////////////////////////////

// (1-alpha)*A + alpha*B. alpha must be between 0~1.
__device__ float lerp(const float A, const float B, float alpha)
{
	return (((1-alpha)*A) + (alpha*B));
}

__global__ void bilinear_resize4x4(unsigned int *out, int width, int height, bool flag) {
	int x, x1, y, y1;	// x1 and y1 for texture wrapping purposes
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int a, b, c, d;
	float x_diff, y_diff, blue, red, green;

	// do the 4x4 bilinear scaling
	if (flag == true) { 
		x = CLAMP((j-2), 0, width-1)>>2;
		y = CLAMP((i-2), 0, height-1)>>2;
		x1 = x+1;
		y1 = y+1;
		x_diff = (CLAMP(j-2, 0, width) - (x*4)) * ONE_FOURTH; // j%4 = j - (x*4)
		y_diff = (CLAMP(i-2, 0, height) - (y*4)) * ONE_FOURTH;
	} else {
		x = (j>>2);
		y = (i>>2);
		x1 = CLAMP(x+1, 0, (width>>2)-1);
		y1 = CLAMP(y+1, 0, (width>>2)-1);
		x_diff = (j - (x*4)) * ONE_FOURTH; // j%4 = j - (x*4)
		y_diff = (i - (y*4)) * ONE_FOURTH;
	}

	// fetch the 32bit ARGB unsigned ints for the four pixels
	a = tex2D(texRef, x, y);
	b = tex2D(texRef, x1, y);
	c = tex2D(texRef, x, y1);
	d = tex2D(texRef, x1, y1);

	// blue channel
	blue = lerp(lerp((float)MAKE_BLUE_PIXEL(a), (float)MAKE_BLUE_PIXEL(b), x_diff),
		lerp((float)MAKE_BLUE_PIXEL(c), (float)MAKE_BLUE_PIXEL(d), x_diff),
		y_diff);

	// green channel
	green = lerp(lerp((float)MAKE_GREEN_PIXEL(a), (float)MAKE_GREEN_PIXEL(b), x_diff),
		lerp((float)MAKE_GREEN_PIXEL(c), (float)MAKE_GREEN_PIXEL(d), x_diff),
		y_diff);

	// red channel
	red = lerp(lerp((float)MAKE_RED_PIXEL(a), (float)MAKE_RED_PIXEL(b), x_diff),
		lerp((float)MAKE_RED_PIXEL(c), (float)MAKE_RED_PIXEL(d), x_diff),
		y_diff);

	// store the result
	out[i*width + j] = MAKE_ARGB( CLAMP(__float2uint_rn(red), 0, 255),
		CLAMP(__float2uint_rn(green), 0, 255),
		CLAMP(__float2uint_rn(blue), 0, 255) );
}

__global__ void bilinear_resize8x4(unsigned int *out, int width, int height, bool flag) {
	int x, x1, y, y1;	// x1 and y1 for texture wrapping purposes
	int j = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int a, b, c, d;
	float x_diff, y_diff, blue, red, green;

	// do the 8x4 bilinear scaling
	if (flag == true) {
		x = CLAMP((j-4), 0, width-1)>>3;
		y = CLAMP((i-2), 0, height-1)>>2;
		x1 = x+1;
		y1 = y+1;
		x_diff = (CLAMP(j-4, 0, width) - (x*8)) * ONE_EIGHTH; // j%4 = j - (x*4)
		y_diff = (CLAMP(i-2, 0, height) - (y*4)) * ONE_FOURTH;
	} else {
		x = (j>>3);
		y = (i>>2);
		x1 = CLAMP(x+1, 0, (width>>3)-1);
		y1 = CLAMP(y+1, 0, (width>>2)-1);
		x_diff = (j - (x*8)) * ONE_EIGHTH; // j%4 = j - (x*4)
		y_diff = (i - (y*4)) * ONE_FOURTH;
	}
//	x = j>>3;		// (j / 8)
//	y = i>>2;		// (i / 4)
//	x1 = (x+1) & ((width>>3) - 1);	// (x+1) % (width / 8)
//	y1 = (y+1) & ((height>>2) - 1);	// (y+1) % (height / 4)
//	x_diff = (j - (x*8)) * ONE_EIGHTH;	// j%8 = j - (x*8)
//	y_diff = (i - (y*4)) * ONE_FOURTH;	// i%4 = j - (y*4)

	// fetch the 32bit ARGB unsigned ints for the four pixels
	a = tex2D(texRef, x, y);
	b = tex2D(texRef, x1, y);
	c = tex2D(texRef, x, y1);
	d = tex2D(texRef, x1, y1);

	// blue channel
	blue = lerp(lerp((float)MAKE_BLUE_PIXEL(a), (float)MAKE_BLUE_PIXEL(b), x_diff),
		lerp((float)MAKE_BLUE_PIXEL(c), (float)MAKE_BLUE_PIXEL(d), x_diff),
		y_diff);

	// green channel
	green = lerp(lerp((float)MAKE_GREEN_PIXEL(a), (float)MAKE_GREEN_PIXEL(b), x_diff),
		lerp((float)MAKE_GREEN_PIXEL(c), (float)MAKE_GREEN_PIXEL(d), x_diff),
		y_diff);

	// red channel
	red = lerp(lerp((float)MAKE_RED_PIXEL(a), (float)MAKE_RED_PIXEL(b), x_diff),
		lerp((float)MAKE_RED_PIXEL(c), (float)MAKE_RED_PIXEL(d), x_diff),
		y_diff);

	// store the result
	out[i*width + j] = MAKE_ARGB(CLAMP(__float2uint_rn(red), 0, 255),
                                 CLAMP(__float2uint_rn(green), 0, 255),
                                 CLAMP(__float2uint_rn(blue), 0, 255));
}
//////////////////////////////////////////////////////////////////////////////////
//								 bilinear upscaling								//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//							low-pass wavelet filter								//
//////////////////////////////////////////////////////////////////////////////////
__global__ void linear_wavelet_transform_rows(int width, int height, 
											  unsigned int* out) {
	int n; 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int pixel;
	float filter;

	float3 s = make_float3(0.0f, 0.0f, 0.0f);

	// we start out negative because we want to center the filter on
	// the current pixel
	for (int m = 0 - HALF_FILTER_LENGTH; m < HALF_FILTER_LENGTH; m++) {
		n = 2 * k + m;
		if (n < 0) {
			n = 0 - n;
		}
		if (n >= width) {
			n -= 2 * (1 + n - width);
		}
		pixel = tex2D(texRef, CLAMP(n, 0, width-1), CLAMP(y, 0, height-1));
		filter = tex1Dfetch(filterRef, (int)(m + HALF_FILTER_LENGTH));

		s.x += filter * __uint2float_rn(MAKE_RED_PIXEL(pixel));
		s.y += filter * __uint2float_rn(MAKE_GREEN_PIXEL(pixel));
		s.z += filter * __uint2float_rn(MAKE_BLUE_PIXEL(pixel));
	}

	out[y*width + k] = MAKE_ARGB(CLAMP(__float2uint_rn(s.x), 0, 255),
								 CLAMP(__float2uint_rn(s.y), 0, 255),
								 CLAMP(__float2uint_rn(s.z), 0, 255));
}

__global__ void linear_wavelet_transform_cols(int width, int height, 
											  unsigned int* out) {
	int n; 
	int k = blockDim.y * blockIdx.y + threadIdx.y;
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int pixel;
	float filter;

	float3 s = make_float3(0.0f, 0.0f, 0.0f);

	// we start out negative because we want to center the filter on
	// the current pixel
	for (int m = 0 - HALF_FILTER_LENGTH; m < HALF_FILTER_LENGTH; m++) {
		n = 2 * k + m;
		if (n < 0) {
			n = 0 - n;
		}
		if (n >= height) {
			n -= 2 * (1 + n - height);
		}
		pixel = tex2D(texRef, CLAMP(x, 0, width-1), CLAMP(n, 0, height-1));
		filter = tex1Dfetch(filterRef, (int)(m + HALF_FILTER_LENGTH));

		s.x += filter * __uint2float_rn(MAKE_RED_PIXEL(pixel));
		s.y += filter * __uint2float_rn(MAKE_GREEN_PIXEL(pixel));
		s.z += filter * __uint2float_rn(MAKE_BLUE_PIXEL(pixel));
	}

	out[k*width + x] = MAKE_ARGB(CLAMP(__float2uint_rn(s.x), 0, 255),
								 CLAMP(__float2uint_rn(s.y), 0, 255),
								 CLAMP(__float2uint_rn(s.z), 0, 255));
}

//////////////////////////////////////////////////////////////////////////////////
//	Calculates the low-pass wavelet transform of the input data and stores the	//
//	result in the same location. First transforms the rows and then the colums	//
//	of the input image.															//
//																				//
//	wavelet		Initial input data. The result is also stored here				//
//	temp		Pointer to scratch space in memory that is the same size as the	//
//				input data														//
//	num			The number of times the low-pass filter is applied				//
//																				//
//////////////////////////////////////////////////////////////////////////////////
void linear_wavelet_transform(unsigned int * wavelet, unsigned int *temp, int num,
							  textureReference *texRefPtr, int width, int height, 
							  int scan_width, hipChannelFormatDesc *channelDesc,
							  dim3 default_block){
	size_t offset;
	int i;
	for (i = 1; i <= num; i++) {
		// bind the input data to the texture
		cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)wavelet, 
			channelDesc, width, height, scan_width));

		// calulate the grid dimensions and call the kernel
		dim3 grid0(width / (BLOCK_WIDTH * 2 * i), height / (BLOCK_HEIGHT * i));
		linear_wavelet_transform_rows<<<grid0, default_block>>> (width, height, temp);

		// bind the new data to the texture
		cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)temp, 
			channelDesc, width, height, scan_width));
		
		// calulate the grid dimensions and call the kernel
		dim3 grid1(width / (BLOCK_WIDTH * i), height / (BLOCK_HEIGHT * 2 * i));
		linear_wavelet_transform_cols<<<grid1, default_block>>> (width, height, wavelet);
	}
}
//////////////////////////////////////////////////////////////////////////////////
//							low-pass wavelet filter								//
//////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////
//	External method to call CUDA kernels for compression. Places the compressed	//
//	data in	h_out.																//
//																				//
//	width		Width of the input image										//
//	height		Height of the input image										//
//	scan_width	Size in bytes of a single row of pixels							//
//	h_in		Pointer to the input buffer on the host							//
//	h_out		Pointer to the output buffer on the	host						//
//																				//
//////////////////////////////////////////////////////////////////////////////////
extern "C" int cuda_pvr_compress(int width, int height, int scan_width, 
								  unsigned char* h_in, unsigned char* h_out) {
	// image dimensions must be powers of 2
	if ((width & (width - 1)) != 0 || (height & (height - 1)) != 0) {
		printf("Error: Image dimensions must be powers of 2.\n");
		printf("Aborting compression...\n");

		return 1;
	}

	// declare device memory pointers and block dimensions
	unsigned int *d_bits,*d_out,*d_temp,*d_wavelet,*d_axis,*d_aproto,*d_bproto;
	int *d_redCurrent, *d_greenCurrent, *d_blueCurrent, *d_err;
	float *d_filter, *d_mod;
#ifdef TWO_BY_TWO
    float *d_svdMatrixTL, *d_svdMatrixTR, *d_svdMatrixBL, *d_svdMatrixBR;
#else
    float *d_svdMatrix;
#endif // TWO_BY_TWO
	int i, tex_size = height * scan_width;
	dim3 default_block(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 default_grid(width / (BLOCK_WIDTH), height / (BLOCK_HEIGHT));
	dim3 quarterGrid(default_grid.x>>2, default_grid.y>>2);
    dim3 svdGrid(default_grid.x * SVD_FACTOR_X, default_grid.y * SVD_FACTOR_Y);
	size_t offset;

	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

	// initialize cuda and allocate device memory
	CUT_DEVICE_INIT(1, "");
	cutilSafeCall(hipMalloc((void **)&d_bits, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_temp, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_wavelet, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_axis, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_aproto, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_bproto, tex_size));
	cutilSafeCall(hipMalloc((void **)&d_redCurrent, width * height * sizeof(int)));
	cutilSafeCall(hipMalloc((void **)&d_greenCurrent, width * height * sizeof(int)));
	cutilSafeCall(hipMalloc((void **)&d_blueCurrent, width * height * sizeof(int)));
	cutilSafeCall(hipMalloc((void **)&d_err, sizeof(int)));
	cutilSafeCall(hipMalloc((void **)&d_filter, FILTER_LENGTH * sizeof(float)));
	cutilSafeCall(hipMalloc((void **)&d_mod, width * height * sizeof(float)));
#ifdef TWO_BPP
    cutilSafeCall(hipMalloc((void **)&d_out, 2 * (width>>3) * (height>>2) * 
                             sizeof(unsigned int)));
#else
	cutilSafeCall(hipMalloc((void **)&d_out, 2 * (width>>2) * (height>>2) * 
		sizeof(unsigned int)));
#endif // TWO_BPP

	cutilSafeCall(hipMemcpy(d_bits, h_in, tex_size, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_wavelet, d_bits, tex_size, hipMemcpyDeviceToDevice));
	cutilSafeCall(hipMemcpy(d_filter, wavelet_filter, FILTER_LENGTH * sizeof(float), 
							 hipMemcpyHostToDevice));
#ifdef TWO_BY_TWO
    cutilSafeCall(hipMalloc((void **)&d_svdMatrixTL, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipMalloc((void **)&d_svdMatrixTR, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipMalloc((void **)&d_svdMatrixBL, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipMalloc((void **)&d_svdMatrixBR, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipMemcpy(d_svdMatrixTL, MwTL, SVD_MAT_HEIGHT * sizeof(float), 
							 hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_svdMatrixTR, MwTR, SVD_MAT_HEIGHT * sizeof(float), 
							 hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_svdMatrixBL, MwBL, SVD_MAT_HEIGHT * sizeof(float), 
							 hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_svdMatrixBR, MwBR, SVD_MAT_HEIGHT * sizeof(float), 
							 hipMemcpyHostToDevice));
#else
    cutilSafeCall(hipMalloc((void **)&d_svdMatrix, SVD_MAT_HEIGHT * sizeof(float)));
	cutilSafeCall(hipMemcpy(d_svdMatrix, Mw, SVD_MAT_HEIGHT * sizeof(float), 
							 hipMemcpyHostToDevice));
#endif // TWO_BY_TWO

	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

	// map the input buffer as texture memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned int>();
	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float>();
	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<int>();

	textureReference *texRefPtr,*origRefPtr,*filterRefPtr,*modRefPtr,
		*redRefPtr,*greenRefPtr,*blueRefPtr; 

	hipGetTextureReference((const textureReference**)&texRefPtr, "texRef");
	hipGetTextureReference((const textureReference**)&origRefPtr, "origRef");
	hipGetTextureReference((const textureReference**)&filterRefPtr, "filterRef");
	hipGetTextureReference((const textureReference**)&modRefPtr, "modRef");
	hipGetTextureReference((const textureReference**)&redRefPtr, "redCurrentRef");
	hipGetTextureReference((const textureReference**)&greenRefPtr, "greenCurrentRef");
	hipGetTextureReference((const textureReference**)&blueRefPtr, "blueCurrentRef");
    
#ifdef TWO_BY_TWO
    textureReference *svdRefPtrTL, *svdRefPtrTR, *svdRefPtrBL, *svdRefPtrBR;
    hipGetTextureReference((const textureReference**)&svdRefPtrTL, "svdMatRefTL");
    hipGetTextureReference((const textureReference**)&svdRefPtrTR, "svdMatRefTR");
    hipGetTextureReference((const textureReference**)&svdRefPtrBL, "svdMatRefBL");
    hipGetTextureReference((const textureReference**)&svdRefPtrBR, "svdMatRefBR");
#else
    textureReference *svdRefPtr;
    hipGetTextureReference((const textureReference**)&svdRefPtr, "svdMatRef");
#endif // TWO_BY_TWO

	// set the addressing and filter modes for all 2D textures
	texRefPtr->addressMode[0] = hipAddressModeClamp;
	texRefPtr->addressMode[1] = hipAddressModeClamp;
	texRefPtr->addressMode[2] = hipAddressModeClamp;
	texRefPtr->filterMode = hipFilterModePoint;

	origRefPtr->addressMode[0] = hipAddressModeClamp;
	origRefPtr->addressMode[1] = hipAddressModeClamp;
	origRefPtr->addressMode[2] = hipAddressModeClamp;
	origRefPtr->filterMode = hipFilterModePoint;

	modRefPtr->addressMode[0] = hipAddressModeClamp;
	modRefPtr->addressMode[1] = hipAddressModeClamp;
	modRefPtr->addressMode[2] = hipAddressModeClamp;
	modRefPtr->filterMode = hipFilterModePoint;

	redRefPtr->addressMode[0] = hipAddressModeClamp;
	redRefPtr->addressMode[1] = hipAddressModeClamp;
	redRefPtr->addressMode[2] = hipAddressModeClamp;
	redRefPtr->filterMode = hipFilterModePoint;

	blueRefPtr->addressMode[0] = hipAddressModeClamp;
	blueRefPtr->addressMode[1] = hipAddressModeClamp;
	blueRefPtr->addressMode[2] = hipAddressModeClamp;
	blueRefPtr->filterMode = hipFilterModePoint;

	greenRefPtr->addressMode[0] = hipAddressModeClamp;
	greenRefPtr->addressMode[1] = hipAddressModeClamp;
	greenRefPtr->addressMode[2] = hipAddressModeClamp;
	greenRefPtr->filterMode = hipFilterModePoint;

	// bind textures
	cutilSafeCall(hipBindTexture2D(&offset, origRefPtr, (const void*)d_bits, 
		&channelDesc, width, height, scan_width));
	cutilSafeCall(hipBindTexture(&offset, filterRefPtr, (const void*)d_filter, 
		&channelDesc2, FILTER_LENGTH * sizeof(float)));
	cutilSafeCall(hipBindTexture2D(&offset, modRefPtr, (const void*)d_mod, 
		&channelDesc2, width, height, width * sizeof(float)));
	cutilSafeCall(hipBindTexture2D(&offset, redRefPtr, (const void*)d_redCurrent, 
		&channelDesc3, width, height, width * sizeof(int)));
	cutilSafeCall(hipBindTexture2D(&offset, greenRefPtr, (const void*)d_greenCurrent, 
		&channelDesc3, width, height, width * sizeof(int)));
	cutilSafeCall(hipBindTexture2D(&offset, blueRefPtr, (const void*)d_blueCurrent, 
		&channelDesc3, width, height, width * sizeof(int)));
#ifdef TWO_BY_TWO
    cutilSafeCall(hipBindTexture(&offset, svdRefPtrTL, (const void*)d_svdMatrixTL, 
                                  &channelDesc2, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipBindTexture(&offset, svdRefPtrTR, (const void*)d_svdMatrixTR, 
                                  &channelDesc2, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipBindTexture(&offset, svdRefPtrBL, (const void*)d_svdMatrixBL, 
                                  &channelDesc2, SVD_MAT_HEIGHT * sizeof(float)));
    cutilSafeCall(hipBindTexture(&offset, svdRefPtrBR, (const void*)d_svdMatrixBR, 
                                  &channelDesc2, SVD_MAT_HEIGHT * sizeof(float)));
#else
    cutilSafeCall(hipBindTexture(&offset, svdRefPtr, (const void*)d_svdMatrix, 
                                  &channelDesc2, SVD_MAT_HEIGHT * sizeof(float)));
#endif // TWO_BY_TWO

	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

	// apply low-pass filter to initial image
	linear_wavelet_transform(d_wavelet, d_temp, 2, texRefPtr, width, height, 
							 scan_width, &channelDesc, default_block);
	
	// bilinear upscale
	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_wavelet, 
				  &channelDesc, width, height, scan_width));
#ifdef TWO_BPP
    dim3 grid0(default_grid.x>>3, default_grid.y>>2);
    // one more time for 8x4
    linear_wavelet_transform_rows<<<grid0, default_block>>> (width, height, d_temp);
    
    cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_temp, 
                                    &channelDesc, width, height, scan_width));
    bilinear_resize8x4<<<default_grid, default_block>>> (d_wavelet, width, height, false);
    //printf("Finished low-pass filter...\n");
#else
	bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, false);
	swap((void**)&d_temp, (void**)&d_wavelet);
#endif // TWO_BPP
	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

	// make the inital A and B prototypes
	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_wavelet, 
				  &channelDesc, width, height, scan_width));
	make_a_b_prototypes<<<default_grid, default_block>>> (d_aproto, d_bproto, 
		width, height);

    //printf("Made initial A and B prototypes...\n");
    
	// apply low-pass filter to A and B prototypes to get initial candidates
	linear_wavelet_transform(d_aproto, d_temp, 2, texRefPtr, width, height, 
		scan_width, &channelDesc, default_block);
    linear_wavelet_transform(d_bproto, d_temp, 2, texRefPtr, width, height, 
                             scan_width, &channelDesc, default_block);
    
#ifdef TWO_BPP
    // one more time for 8x4
    cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_aproto, 
                                    &channelDesc, width, height, scan_width));
    linear_wavelet_transform_rows<<<grid0, default_block>>> (width, height, d_temp);
    swap((void**)&d_temp, (void**)&d_aproto);
    
    cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_bproto, 
                                    &channelDesc, width, height, scan_width));
    linear_wavelet_transform_rows<<<grid0, default_block>>> (width, height, d_temp);
    swap((void**)&d_temp, (void**)&d_bproto);
    //printf("Finished low-pass filter for A and B...\n");
#endif   // TWO_BPP
    
	// copy the prototypes into temporary storage (re-using d_axis and d_wavelet
	// for temporary storage)
	hipMemcpy(d_axis, d_aproto, tex_size, hipMemcpyDeviceToDevice);
	hipMemcpy(d_wavelet, d_bproto, tex_size, hipMemcpyDeviceToDevice);

	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

	// optimize A and B candidates and encode final image
	for (i = 0; i < NUM_OPTIMIZATION_PASSES; i++) {
		// expand the A and B candidates to full resolution
		cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_aproto, 
			&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
        bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
		bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
		swap((void**)&d_temp, (void**)&d_aproto);

		cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_bproto, 
			&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
        bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
		bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
		swap((void**)&d_temp, (void**)&d_bproto);

		// get the modulation bits 
		compute_modulation_bits<<<default_grid, default_block>>> (d_aproto, d_bproto, 
			d_redCurrent, d_greenCurrent, d_blueCurrent, d_mod, width);

#ifdef TWO_BPP
        compute_modulation_mode<<<grid0, default_block>>> (d_aproto, d_bproto, d_temp,
            d_mod, d_redCurrent, d_greenCurrent, d_blueCurrent, width);
#endif // TWO_BPP
		// make optimization call
		cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_bits, 
										&channelDesc, width, height, scan_width));
#ifdef USE_SVD
		svd_optimize<<<svdGrid, default_block>>>(d_aproto, d_bproto, d_axis,
			d_wavelet, width, height, d_err);
#else
#ifdef USE_JAMA_SVD
        svd_optimize<<<svdGrid, default_block>>>(d_aproto, d_bproto, d_axis,
			d_wavelet, width, height, d_err);
#else
#ifdef USE_CHOLESKY
        cholesky_optimize<<<svdGrid, default_block>>>(d_aproto, d_bproto, d_axis,
            d_wavelet, width, height, d_err);
#else
		moore_penrose_optimize<<<quarterGrid, default_block>>>(d_aproto, d_bproto, d_axis,
			d_wavelet, width, height, d_err);
#endif // USE_CHOLESKY  
#endif // USE_JAMA_SVD
#endif // USE_SVD
	}

	// check for SVD error
	int kernelError;
	hipMemcpy(&kernelError, d_err, sizeof(int), hipMemcpyDeviceToHost);
	if (kernelError == 1) {
//		printf("One or more SVD matrices did not converge in 30 iterations.\n");
//		printf("Compression failed.\n");
//		printf("Please try again.\n\n");
//		printf("Press ENTER to exit.\n");
//		getchar();
//		exit(EXIT_FAILURE);
        return EXIT_FAILURE;
	}

    //printf("Finished optimization passes...\n");
	//////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////

#ifdef DECOMPRESS_PVR
	// expand the A and B candidates to full resolution
	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_aproto, 
		&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
    bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
    bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
	swap((void**)&d_temp, (void**)&d_aproto);

	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_bproto, 
		&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
    bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
    bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
	swap((void**)&d_temp, (void**)&d_bproto);

	decompress<<<default_grid, default_block>>>(d_aproto, d_bproto, d_wavelet, d_mod, width);
	cutilSafeCall(hipMemcpy(h_out, d_wavelet, tex_size, hipMemcpyDeviceToHost));
#else

	// return compressed data
    //printf("Encoding Texture...\n");
#ifdef TWO_BPP
    encode_texture<<<grid0, default_block>>>(d_aproto, d_bproto, d_mod, width, 
        false, d_out, d_temp, width>>3, height>>2);
	cutilSafeCall(hipMemcpy(h_out, d_out, 2 * (width>>3) * (height>>2) * 
                             sizeof(unsigned int), hipMemcpyDeviceToHost));
#else
	encode_texture<<<quarterGrid, default_block>>>(d_aproto, d_bproto, d_mod, width, 
		false, d_out, d_temp, width>>2, height>>2);
	cutilSafeCall(hipMemcpy(h_out, d_out, 2 * (width>>2) * (height>>2) * 
		sizeof(unsigned int), hipMemcpyDeviceToHost));
	//cutilSafeCall(hipMemcpy(h_out, d_aproto, tex_size, hipMemcpyDeviceToHost));
#endif  // TWO_BPP
    //printf("Finished copying back texture...\n");
    
#ifdef GET_RMS_ERROR
	// expand the A and B candidates to full resolution
	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_aproto, 
		&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
    bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
    bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
	swap((void**)&d_temp, (void**)&d_aproto);

	cutilSafeCall(hipBindTexture2D(&offset, texRefPtr, (const void*)d_bproto, 
		&channelDesc, width, height, scan_width));
#ifdef TWO_BPP
    bilinear_resize8x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#else
    bilinear_resize4x4<<<default_grid, default_block>>> (d_temp, width, height, true);
#endif // TWO_BPP
	swap((void**)&d_temp, (void**)&d_bproto);

	// get the per pixel error
	unsigned int *h_rms, *d_rms, j;
	float error, mean_error, total_error = 0.0f;
	h_rms = (unsigned int*)malloc(width * height * sizeof(int));
	cutilSafeCall(hipMalloc((void **)&d_rms, width * height * sizeof(int)));

	rms_error<<<default_grid, default_block>>>(d_bits, d_aproto, d_bproto, d_rms, width);
	cutilSafeCall(hipMemcpy(h_rms, d_rms, width * height * sizeof(int), 
		hipMemcpyDeviceToHost));
	hipDeviceSynchronize();
	for (i = 0; i < height; i++) {
		for (j = 0; j < width; j++) {
			total_error += h_rms[i*width + j];
		}
	}
	mean_error = total_error / (width * height * 3);
	error = sqrtf(mean_error);
	printf("RMS Error is %.3f.\n", error);
#ifdef GET_SNR
	printf("Peak signal-to-noise ratio is %0.3fdB.\n", 10 * log10(SQR(255) / mean_error));
#endif // GET_SNR
#endif // GET_RMS_ERROR
#endif // DECOMPRESS_PVR

	// cleanup
	hipDeviceReset();

	return EXIT_SUCCESS;
}